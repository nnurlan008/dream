#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <string>
#include <cstring>

#include <chrono>

using namespace std;
// using namespace std;


// extern "C"{
//   #include "rdma_utils.h"
// }

// #include "../../src/rdma_utils.cuh"
#include <time.h>
// #include "../../include/runtime_prefetching.h"
#include "../../include/runtime_eviction.h"
// #include "../../include/runtime_prefetching_2nic.h"

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.05

//define a small float value
#define SMALL_FLOAT_VAL 0.00000001f

// Size of array
#define N 4096*16llu

#define BLOCK_NUM 1024ULL
#define MYINFINITY 2147483647llu

#define BLOCK_SIZE 1024
#define WARP_SHIFT 5
#define WARP_SIZE 32


#define GPU 0

typedef float DATA_TYPE;

/* Thread block dimensions */
#define DIM_THREAD_BLOCK_X 1024
#define DIM_THREAD_BLOCK_Y 1

__device__ rdma_buf<unsigned int> D_adjacencyList;

__global__ void test(rdma_buf<unsigned int> *a/*, rdma_buf<int> *b, rdma_buf<int> *c*/);


// Kernel
__global__ void add_vectors_uvm(int *a, int *b, int *c, int size)
{
	int id = blockDim.x * blockIdx.x + threadIdx.x;
	// if(id < size) {
		c[id] = a[id] + b[id];
		// printf("c[%d]: %d\n", id, c[id]);
	// }
}

#define htonl(x)  ((((uint32_t)(x) & 0xff000000) >> 24) |\
                   (((uint32_t)(x) & 0x00ff0000) >>  8) |\
                   (((uint32_t)(x) & 0x0000ff00) <<  8) |\
                   (((uint32_t)(x) & 0x000000ff) << 24))

#define WARP_SIZE 32

void delay(int number_of_seconds)
{
    // Converting time into milli_seconds
    int milli_seconds = 1000000 * number_of_seconds;
 
    // Storing start time
    clock_t start_time = clock();
 
    // looping till required time is not achieved
    while (clock() < start_time + milli_seconds)
        ;
}

enum { NS_PER_SECOND = 1000000000 };

void sub_timespec(struct timespec t1, struct timespec t2, struct timespec *td)
{
    td->tv_nsec = t2.tv_nsec - t1.tv_nsec;
    td->tv_sec  = t2.tv_sec - t1.tv_sec;
    if (td->tv_sec > 0 && td->tv_nsec < 0)
    {
        td->tv_nsec += NS_PER_SECOND;
        td->tv_sec--;
    }
    else if (td->tv_sec < 0 && td->tv_nsec > 0)
    {
        td->tv_nsec -= NS_PER_SECOND;
        td->tv_sec++;
    }
}

void usage(const char *argv0)
{
  fprintf(stderr, "usage: %s <mode> <server-address> <server-port>\n  mode = \"read\", \"write\"\n", argv0);
  exit(1);
}


#define check_cuda_error(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }

}

void init_array(DATA_TYPE* A, DATA_TYPE* x1, DATA_TYPE* x2, DATA_TYPE* y1, DATA_TYPE* y2)
{
	size_t i, j;

	for (i = 0; i < N; i++)
	{
		x1[i] = ((DATA_TYPE) i) / N;
		x2[i] = ((DATA_TYPE) i + 1) / N;
		y1[i] = ((DATA_TYPE) i + 3) / N;
		y2[i] = ((DATA_TYPE) i + 4) / N;
		// x1_gpu[i] = ((DATA_TYPE) i) / N;
		// x2_gpu[i] = ((DATA_TYPE) i + 1) / N;
		// y_1_gpu[i] = ((DATA_TYPE) i + 3) / N;
		// y_2_gpu[i] = ((DATA_TYPE) i + 4) / N;
		for (j = 0; j < N; j++)
		{
			A[i*N + j] = ((DATA_TYPE) i*j) / N;
			// a_gpu[i*N + j] = ((DATA_TYPE) i*j) / N;
		}
	}
}

__global__ void transfer(size_t size, rdma_buf<DATA_TYPE> *d_adjacencyList)
{
    size_t id = blockDim.x * blockIdx.x + threadIdx.x;
    size_t stride = blockDim.x * gridDim.x;
    
        for (size_t i = id; i < size ; i += stride)
        {
            DATA_TYPE y = (*d_adjacencyList)[i];
        }
}

__global__ void check(size_t size, rdma_buf<DATA_TYPE> *d_adjacencyList, DATA_TYPE *a)
{
    size_t id = blockDim.x * blockIdx.x + threadIdx.x;
    size_t stride = blockDim.x * gridDim.x;
    
        for (size_t i = id; i < size ; i += stride)
        {
            DATA_TYPE y = (*d_adjacencyList)[i];
            if(a[i] != y){
                printf("y: %f %f ", y, a[i]);
            }
        }
}

__global__ void assign_array(rdma_buf<unsigned int> *adjacencyList){
    D_adjacencyList = *adjacencyList;
    printf("D_adjacencyList.d_TLB[0].state: %d\n", D_adjacencyList.d_TLB[0].state);
    printf("D_adjacencyList.d_TLB[0].device_address: %p\n", D_adjacencyList.d_TLB[0].device_address);
}

int alloc_global_cont(struct post_content *post_cont, struct poll_content *poll_cont, struct post_content2 *post_cont2){
    struct post_content *d_post;
    struct poll_content *d_poll;
    struct post_content2 *d_post2;

    hipError_t ret0 = hipMalloc((void **)&d_post, sizeof(struct post_content));
    if(ret0 != hipSuccess){
        printf("Error on allocation post content!\n");
        return -1;
    }
    ret0 = hipMalloc((void **)&d_poll, sizeof(struct poll_content));
    if(ret0 != hipSuccess){
        printf("Error on allocation poll content!\n");
        return -1;
    }
    printf("sizeof(struct post_content): %d, sizeof(struct poll_content): %d\n", sizeof(struct post_content), sizeof(struct poll_content));
    ret0 = hipMemcpy(d_post, post_cont, sizeof(struct post_content), hipMemcpyHostToDevice);
    if(ret0 != hipSuccess){
        printf("Error on post copy!\n");
        return -1;
    }
    ret0 = hipMemcpy(d_poll, poll_cont, sizeof(struct poll_content), hipMemcpyHostToDevice);
    if(ret0 != hipSuccess){
        printf("Error on poll copy!\n");
        return -1;
    }

    ret0 = hipMalloc((void **)&d_post2, sizeof(struct post_content2));
    if(ret0 != hipSuccess){
        printf("Error on allocation post content!\n");
        return -1;
    }
    ret0 = hipMemcpy(d_post2, post_cont2, sizeof(struct post_content2), hipMemcpyHostToDevice);
    if(ret0 != hipSuccess){
        printf("Error on poll copy!\n");
        return -1;
    }

    alloc_content<<<1,1>>>(d_post, d_poll);
    alloc_global_content<<<1,1>>>(d_post, d_poll, d_post2);
    ret0 = hipDeviceSynchronize();
    if(ret0 != hipSuccess){
        printf("Error on alloc_content!\n");
        return -1;
    }
    return 0;
}

__device__ size_t sum_page_faults = 0;

__global__ void
print_retires(void){
    // size_t max = cq_wait[0];
    // for (size_t i = 0; i < 128; i++)
    // {
    //     if(max < cq_wait[i]) max = cq_wait[i];
    // }
    sum_page_faults += g_qp_index;
    printf("g_qp_index: %llu sum page fault: %llu\n", g_qp_index, sum_page_faults);
    g_qp_index = 0;
    // for (size_t i = 0; i < 128; i++)
    // {
    //     max = 0;
    // }
}

void oversubs(float os, size_t size){
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, 0);
    // Calculate memory utilization
    size_t totalMemory = devProp.totalGlobalMem;
    size_t freeMemory;
    size_t usedMemory;
    float workload_size = ((float) size);
    hipMemGetInfo(&freeMemory, &totalMemory);
    usedMemory = totalMemory - freeMemory;
    printf("Total GPU Memory: %.2f MiB\n", (float) totalMemory / (1024 * 1024));
    printf("Free GPU Memory: %.2f MiB\n", (float) freeMemory / (1024 * 1024));
    printf("Used GPU Memory: %.2f MiB\n", (float) usedMemory / (1024 * 1024));

    printf("Workload size: %.2f\n", workload_size/1024/1024);
    float oversubs_ratio = (float) os;
    void *tmp_ptr;
    hipMalloc(&tmp_ptr, (size_t) (freeMemory - workload_size));
    hipMemGetInfo(&freeMemory, &totalMemory);
    printf("Free GPU Memory: %.2f MiB\n", (float) freeMemory / (1024 * 1024));
    if(oversubs_ratio > 0){
        
        void *over_ptr;
        long long unsigned int os_size = freeMemory - workload_size /(1 + oversubs_ratio);
        printf("workload: %.2f\n",  workload_size);
        printf("workload: %llu\n",  os_size);
        hipMalloc(&over_ptr, os_size); 
        printf("os_size: %u\n", os_size/1024/1024);
    }
    hipMemGetInfo(&freeMemory, &totalMemory);
    printf("Free GPU Memory: %.2f MiB\n", (float) freeMemory / (1024 * 1024));
}

__global__
void print_utilization() {
    printf("GPU_address_offset: %llu \n", GPU_address_offset);
}

float absVal(float a)
{
	if(a < 0)
	{
		return (a * -1);
	}
   	else
	{ 
		return a;
	}
}


float percentDiff(double val1, double val2)
{
	if ((absVal(val1) < 0.01) && (absVal(val2) < 0.01))
	{
		return 0.0f;
	}

	else
	{
    		return 100.0f * (absVal(absVal(val1 - val2) / absVal(val1 + SMALL_FLOAT_VAL)));
	}
} 

void runMvt(DATA_TYPE* a, DATA_TYPE* x1, DATA_TYPE* x2, DATA_TYPE* y1, DATA_TYPE* y2)
{
	size_t i, j;
	
	for (i=0; i<N; i++) 
	{
		for (j=0; j<N; j++) 
		{
       			x1[i] = x1[i] + a[i*N + j] * y1[j];
        	}
    	}
	
	for (i=0; i<N; i++) 
	{
		for (j=0; j<N; j++) 
		{
 		       	x2[i] = x2[i] + a[j*N + i] * y2[j];
      		}
    	}
}


void compareResults(DATA_TYPE* x1, DATA_TYPE* x1_outputFromGpu, DATA_TYPE* x2, DATA_TYPE* x2_outputFromGpu)
{
	size_t i, fail;
	fail = 0;
	
	for (i=0; i<N; i++) 
	{
		if (percentDiff(x1[i], x1_outputFromGpu[i]) > PERCENT_DIFF_ERROR_THRESHOLD)
		{
            // printf("x1: %f  x1_uvm: %f\n", x1, x1_outputFromGpu);
			fail++;
		}

		if (percentDiff(x2[i], x2_outputFromGpu[i]) > PERCENT_DIFF_ERROR_THRESHOLD)
		{
            // printf("x2: %f  x2_uvm: %f\n", x2[i], x2_outputFromGpu[i]);
			fail++;
		}
	}
	
	// Print results
	printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %llu\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}

/******************************* CUDA Imlementation BEGIN ***************************************/
__global__ void mvt_kernel1(DATA_TYPE *a, DATA_TYPE *x1, DATA_TYPE *y_1)
{
	size_t i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < N)
	{
		size_t j;
		for(j=0; j < N; j++)
		{
			x1[i] += a[i * N + j] * y_1[j];
		}
	}
}


__global__ void mvt_kernel2(DATA_TYPE *a, DATA_TYPE *x2, DATA_TYPE *y_2, DATA_TYPE *a_gpu)
{
	size_t i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < N)
	{
		size_t j;
		for(j=0; j < N; j++)
		{
            size_t index = j * N + i;
            DATA_TYPE tmp = a[index]; 
			// x2[i] += tmp * y_2[j];
            if(tmp != a_gpu[index]){
                printf("tmp: %f %f ", tmp, a_gpu[index]);
            }	
			x2[i] += a_gpu[index] * y_2[j];	
		}
	}
}

void mvtCuda(DATA_TYPE* a, DATA_TYPE* &x1, DATA_TYPE* &x2, DATA_TYPE* y_1, DATA_TYPE* y_2)
{
    
	DATA_TYPE* a_gpu;
	DATA_TYPE* x1_gpu;
	DATA_TYPE* x2_gpu;
	DATA_TYPE* y_1_gpu;
	DATA_TYPE* y_2_gpu;
	DATA_TYPE* x1_cpu;
	DATA_TYPE* x2_cpu;

    x1_cpu = (DATA_TYPE*)malloc(N*sizeof(DATA_TYPE));
	x2_cpu = (DATA_TYPE*)malloc(N*sizeof(DATA_TYPE));

    memcpy(x1_cpu, x1, N*sizeof(DATA_TYPE));
    memcpy(x2_cpu, x2, N*sizeof(DATA_TYPE));
    printf("Allocating on GPU\n");
	
	check_cuda_error(hipMalloc(&x1_gpu, sizeof(DATA_TYPE) * N));   
	check_cuda_error(hipMalloc(&x2_gpu, sizeof(DATA_TYPE) * N));   
	check_cuda_error(hipMalloc(&y_1_gpu, sizeof(DATA_TYPE) * N));  
	check_cuda_error(hipMalloc(&y_2_gpu, sizeof(DATA_TYPE) * N));  
    
    printf("Initializing finished on GPU. Transferring to GPU...\n");
	check_cuda_error(hipMemcpy(y_1_gpu, y_1, sizeof(DATA_TYPE) * N, hipMemcpyHostToDevice));  
	check_cuda_error(hipMemcpy(y_2_gpu, y_2, sizeof(DATA_TYPE) * N, hipMemcpyHostToDevice));  
    check_cuda_error(hipMemcpy(x1_gpu, x1, sizeof(DATA_TYPE) * N, hipMemcpyHostToDevice));  
	check_cuda_error(hipMemcpy(x2_gpu, x2, sizeof(DATA_TYPE) * N, hipMemcpyHostToDevice));  

    bool uvm = true;
    if(uvm) {
        printf("UVM in action\n");
        check_cuda_error(hipMallocManaged(&a_gpu, sizeof(DATA_TYPE) * N * N));
        
        memcpy(a_gpu, a, sizeof(DATA_TYPE) * N * N);

        check_cuda_error(hipMemAdvise(a_gpu, sizeof(DATA_TYPE) * N * N, hipMemAdviseSetReadMostly, 0));
        // check_cuda_error(hipMemAdvise(a_gpu, sizeof(DATA_TYPE) * N * N, hipMemAdviseSetAccessedBy, 0));

        // oversubs(0.33, sizeof(DATA_TYPE) * N * N);
    }
    else{
        check_cuda_error(hipMalloc(&a_gpu, sizeof(DATA_TYPE) * N * N));
        check_cuda_error(hipMemcpy(a_gpu, a, sizeof(DATA_TYPE) * N * N, hipMemcpyHostToDevice));
    }
    DATA_TYPE* a_direct;
    check_cuda_error(hipMalloc(&a_direct, sizeof(DATA_TYPE) * N * N));
    check_cuda_error(hipMemcpy(a_direct, a, sizeof(DATA_TYPE) * N * N, hipMemcpyHostToDevice));

	double t_start, t_end;
	dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
	dim3 grid((size_t)ceil((float)N/ ((float)DIM_THREAD_BLOCK_X)), 1);

    printf("Starting Kernels\n");
	auto start = std::chrono::steady_clock::now();
	mvt_kernel1<<<grid,block>>>(a_gpu,x1_gpu,y_1_gpu);
	mvt_kernel2<<<grid,block>>>(a_gpu,x2_gpu,y_2_gpu, a_direct);
	hipDeviceSynchronize();
	auto end = std::chrono::steady_clock::now();
    long duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
    printf("Elapsed time for normal cuda in milliseconds: %li ms.d\n\n", duration);

    check_cuda_error(hipMemcpy(x1, x1_gpu, sizeof(DATA_TYPE) * N, hipMemcpyDeviceToHost));   
	check_cuda_error(hipMemcpy(x2, x2_gpu, sizeof(DATA_TYPE) * N, hipMemcpyDeviceToHost)); 

    check_cuda_error(hipFree(a_gpu));
    check_cuda_error(hipFree(x1_gpu));
    check_cuda_error(hipFree(x2_gpu));
    check_cuda_error(hipFree(y_1_gpu));
    check_cuda_error(hipFree(y_2_gpu));

    check_cuda_error(hipFree(a_direct));

    // //run the algorithm on the CPU
    // printf("Running on CPU\n");
	// runMvt(a, x1_cpu, x2_cpu, y_1, y_2);  
    // printf("Comparing Results for CPU and Direct transfer\n");
    // compareResults(x1_cpu, x1, x2_cpu, x2);


}
/******************************* CUDA Imlementation END ***************************************/



/******************************* RDMA Imlementation BEGIN ***************************************/
__global__ void mvt_kernel1_rdma(rdma_buf<DATA_TYPE> *a, DATA_TYPE *x1, DATA_TYPE *y_1)
{
	size_t i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < N)
	{
		size_t j;
		for(j=0; j < N; j++)
		{
			x1[i] += (*a)[i * N + j] * y_1[j];
		}
	}
}


__global__ void mvt_kernel2_rdma(rdma_buf<DATA_TYPE> *a, DATA_TYPE *x2, DATA_TYPE *y_2, DATA_TYPE *a_gpu)
{
	size_t i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < N)
	{
		size_t j;
		for(j=0; j < N; j++)
		{
            size_t index = j * N + i;
            DATA_TYPE tmp = (*a)[index]; 
			// x2[i] += tmp * y_2[j];
            // if(tmp != a_gpu[index]){
            //     printf("tmp: %f %f ", tmp, a_gpu[index]);
            // }	
            x2[i] += tmp * y_2[j];
		}
	}
}

void mvtCuda_rdma(DATA_TYPE* a, DATA_TYPE* &x1, DATA_TYPE* &x2, DATA_TYPE* y_1, DATA_TYPE* y_2)
{
    // DATA_TYPE* a_gpu;
	DATA_TYPE* x1_gpu;
	DATA_TYPE* x2_gpu;
	DATA_TYPE* y_1_gpu;
	DATA_TYPE* y_2_gpu;
	DATA_TYPE* x1_cpu;
	DATA_TYPE* x2_cpu;

    x1_cpu = (DATA_TYPE*)malloc(N*sizeof(DATA_TYPE));
	x2_cpu = (DATA_TYPE*)malloc(N*sizeof(DATA_TYPE));
    printf("Allocating on GPU\n");
    
    memcpy(x1_cpu, x1, N*sizeof(DATA_TYPE));
    memcpy(x2_cpu, x2, N*sizeof(DATA_TYPE));
    
	// check_cuda_error(hipMalloc(&a_gpu, sizeof(DATA_TYPE) * N * N));
	check_cuda_error(hipMalloc(&x1_gpu, sizeof(DATA_TYPE) * N));   
	check_cuda_error(hipMalloc(&x2_gpu, sizeof(DATA_TYPE) * N));   
	check_cuda_error(hipMalloc(&y_1_gpu, sizeof(DATA_TYPE) * N));  
	check_cuda_error(hipMalloc(&y_2_gpu, sizeof(DATA_TYPE) * N));

    printf("Initializing finished on GPU. Transferring to GPU...\n"); 
	// check_cuda_error(hipMemcpy(a_gpu, a, sizeof(DATA_TYPE) * N * N, hipMemcpyHostToDevice));
	check_cuda_error(hipMemcpy(y_1_gpu, y_1, sizeof(DATA_TYPE) * N, hipMemcpyHostToDevice));  
	check_cuda_error(hipMemcpy(y_2_gpu, y_2, sizeof(DATA_TYPE) * N, hipMemcpyHostToDevice));  
    check_cuda_error(hipMemcpy(x1_gpu, x1, sizeof(DATA_TYPE) * N, hipMemcpyHostToDevice));  
	check_cuda_error(hipMemcpy(x2_gpu, x2, sizeof(DATA_TYPE) * N, hipMemcpyHostToDevice));    

	dim3 block(DIM_THREAD_BLOCK_X/2, DIM_THREAD_BLOCK_Y);
	dim3 grid((size_t)ceil((float)N/ ((float)DIM_THREAD_BLOCK_X/2)), 1);
    rdma_buf<DATA_TYPE> *rdma_a;

    check_cuda_error(hipMallocManaged((void **) &rdma_a, sizeof(rdma_buf<unsigned int>)));
    
    rdma_a->start(N*N*sizeof(DATA_TYPE));

    for(size_t i = 0; i < N*N; i++){
        rdma_a->local_buffer[i] = a[i];
    }

    // transfer<<<2048, 512>>>(rdma_a->size/sizeof(DATA_TYPE), rdma_a);
    hipError_t ret = hipDeviceSynchronize();
    // check<<<2048, 512>>>(rdma_a->size/sizeof(DATA_TYPE), rdma_a, a_gpu);
    printf("ret: %d hipGetLastError: %d for transfer\n", ret, hipGetLastError());

    printf("Starting Kernels\n");
	auto start = std::chrono::steady_clock::now();
	mvt_kernel1_rdma<<<grid,block>>>(rdma_a, x1_gpu, y_1_gpu);
    printf("ret: %d hipGetLastError: %d for kernel1\n", ret, hipGetLastError());
    mvt_kernel2_rdma<<<grid,block>>>(rdma_a, x2_gpu, y_2_gpu, y_2_gpu);
	ret = hipDeviceSynchronize();
    printf("ret: %d hipGetLastError: %d\n", ret, hipGetLastError());
	auto end = std::chrono::steady_clock::now();
    long duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
    
    printf("Elapsed time for GPU RDMA in milliseconds: %li ms.d\n\n", duration);

    ret = hipDeviceSynchronize();
    print_retires<<<1,1>>>();

    check_cuda_error(hipMemcpy(x1, x1_gpu, sizeof(DATA_TYPE) * N, hipMemcpyDeviceToHost));   
	check_cuda_error(hipMemcpy(x2, x2_gpu, sizeof(DATA_TYPE) * N, hipMemcpyDeviceToHost));

    // check_cuda_error(hipFree(a_gpu));
    check_cuda_error(hipFree(x1_gpu));
    check_cuda_error(hipFree(x2_gpu));
    check_cuda_error(hipFree(y_1_gpu));
    check_cuda_error(hipFree(y_2_gpu));

    // start = std::chrono::steady_clock::now();
    // //run the algorithm on the CPU
    // printf("Running on CPU\n");
	// runMvt(a, x1_cpu, x2_cpu, y_1, y_2); 
    // end = std::chrono::steady_clock::now();
    // duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
    
    // printf("Elapsed time for CPU in milliseconds: %li ms.d\n\n", duration); 
    // printf("Comparing Results for RDMA and CPU\n");
    // compareResults(x1_cpu, x1, x2_cpu, x2);

    free(x1_cpu);
    free(x2_cpu);    
}
/******************************* RDMA Imlementation END ***************************************/



// void mvtCuda_rdma(DATA_TYPE* a, DATA_TYPE* &x1, DATA_TYPE* &x2, DATA_TYPE* y_1, DATA_TYPE* y_2)
// {
//     DATA_TYPE* a_gpu;
// 	DATA_TYPE* x1_gpu;
// 	DATA_TYPE* x2_gpu;
// 	DATA_TYPE* y_1_gpu;
// 	DATA_TYPE* y_2_gpu;
//     DATA_TYPE* x1_rdma;
// 	DATA_TYPE* x2_rdma;
//     DATA_TYPE* x1_direct;
// 	DATA_TYPE* x2_direct;

//     x1_rdma = (DATA_TYPE*)malloc(N*sizeof(DATA_TYPE));
//     x2_rdma = (DATA_TYPE*)malloc(N*sizeof(DATA_TYPE));
//     x1_direct = (DATA_TYPE*)malloc(N*sizeof(DATA_TYPE));
//     x2_direct = (DATA_TYPE*)malloc(N*sizeof(DATA_TYPE));
	
//     printf("Allocating on GPU\n");

// 	check_cuda_error(hipMalloc(&a_gpu, sizeof(DATA_TYPE) * N * N));
// 	check_cuda_error(hipMalloc(&x1_gpu, sizeof(DATA_TYPE) * N));   
// 	check_cuda_error(hipMalloc(&x2_gpu, sizeof(DATA_TYPE) * N));   
// 	check_cuda_error(hipMalloc(&y_1_gpu, sizeof(DATA_TYPE) * N));  
// 	check_cuda_error(hipMalloc(&y_2_gpu, sizeof(DATA_TYPE) * N));

//     printf("Initializing finished on GPU. Transferring to GPU...\n"); 
// 	check_cuda_error(hipMemcpy(a_gpu, a, sizeof(DATA_TYPE) * N * N, hipMemcpyHostToDevice));
// 	check_cuda_error(hipMemcpy(y_1_gpu, y_1, sizeof(DATA_TYPE) * N, hipMemcpyHostToDevice));  
// 	check_cuda_error(hipMemcpy(y_2_gpu, y_2, sizeof(DATA_TYPE) * N, hipMemcpyHostToDevice));    

// 	dim3 block(DIM_THREAD_BLOCK_X/2, DIM_THREAD_BLOCK_Y);
// 	dim3 grid((size_t)ceil((float)N/ ((float)DIM_THREAD_BLOCK_X/2)), 1);





//     rdma_buf<DATA_TYPE> *rdma_a;

//     check_cuda_error(hipMallocManaged((void **) &rdma_a, sizeof(rdma_buf<unsigned int>)));
    
//     rdma_a->start(N*N*sizeof(DATA_TYPE));

//     for(size_t i = 0; i < N*N; i++){
//         rdma_a->local_buffer[i] = a[i];
//     }

//     // transfer<<<2048, 512>>>(rdma_a->size/sizeof(DATA_TYPE), rdma_a);
//     hipError_t ret = hipDeviceSynchronize();
//     // check<<<2048, 512>>>(rdma_a->size/sizeof(DATA_TYPE), rdma_a, a_gpu);
//     printf("ret: %d hipGetLastError: %d for transfer\n", ret, hipGetLastError());

//     printf("Starting Kernels\n");
// 	auto start = std::chrono::steady_clock::now();
//     mvt_kernel2_rdma<<<grid,block>>>(rdma_a, x2_gpu, y_2_gpu, a_gpu);
//     printf("ret: %d hipGetLastError: %d for kernel1\n", ret, hipGetLastError());
// 	mvt_kernel1_rdma<<<grid,block>>>(rdma_a, x1_gpu, y_1_gpu);
// 	ret = hipDeviceSynchronize();
//     printf("ret: %d hipGetLastError: %d\n", ret, hipGetLastError());
// 	auto end = std::chrono::steady_clock::now();
//     long duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
    
//     printf("Elapsed time for GPU RDMA in milliseconds: %li ms.d\n\n", duration);

//     ret = hipDeviceSynchronize();
//     print_retires<<<1,1>>>();

//     check_cuda_error(hipMemcpy(x1, x1_gpu, sizeof(DATA_TYPE) * N, hipMemcpyDeviceToHost));   
// 	check_cuda_error(hipMemcpy(x2, x2_gpu, sizeof(DATA_TYPE) * N, hipMemcpyDeviceToHost));

//     check_cuda_error(hipFree(a_gpu));
//     check_cuda_error(hipFree(x1_gpu));
//     check_cuda_error(hipFree(x2_gpu));
//     check_cuda_error(hipFree(y_1_gpu));
//     check_cuda_error(hipFree(y_2_gpu));

//     start = std::chrono::steady_clock::now();
//     //run the algorithm on the CPU
//     printf("Running on CPU\n");
// 	runMvt(a, x1_cpu, x2_cpu, y_1, y_2); 
//     end = std::chrono::steady_clock::now();
//     duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
    
//     printf("Elapsed time for CPU in milliseconds: %li ms.d\n\n", duration); 
//     printf("Comparing Results for RDMA and CPU\n");
//     compareResults(x1_cpu, x1, x2_cpu, x2);

//     free(x1_cpu);
//     free(x2_cpu);    
// }


// Main program
int main(int argc, char **argv)
{   
    init_gpu(0);

    bool rdma_flag = true;
    struct context *s_ctx = (struct context *)malloc(sizeof(struct context));
    hipError_t ret1;
    if(rdma_flag){
        init_gpu(0);
        int num_msg = (unsigned long) atoi(argv[4]);
        int mesg_size = (unsigned long) atoi(argv[5]);
        int num_bufs = (unsigned long) atoi(argv[6]);

        
        struct post_content post_cont, *d_post, host_post;
        struct poll_content poll_cont, *d_poll, host_poll;
        struct post_content2 post_cont2, *d_post2;
        struct host_keys keys;

        int num_iteration = num_msg;
        s_ctx->n_bufs = num_bufs;

        s_ctx->gpu_buf_size = 16*1024*1024*1024llu; // N*sizeof(int)*3llu;

        // // remote connection:
        // int ret = connect(argv[2], s_ctx);

        // local connect
        char *mlx_name = "mlx5_0";
        int ret = local_connect(mlx_name, s_ctx);

        ret = prepare_post_poll_content(s_ctx, &post_cont, &poll_cont, &post_cont2, \
                                        &host_post, &host_poll, &keys);
        if(ret == -1) {
            printf("Post and poll contect creation failed\n");    
            exit(-1);
        }

        printf("alloc synDev ret: %d\n", hipDeviceSynchronize());
        alloc_global_cont(&post_cont, &poll_cont, &post_cont2);
        // if(hipSuccess != ){    
        printf("alloc synDev ret1: %d\n", hipDeviceSynchronize());
            // return -1;
        // }

        ret1 = hipDeviceSynchronize();
        printf("ret: %d\n", ret1);
        if(hipSuccess != ret1){    
            return -1;
        }

        size_t restricted_gpu_mem = 16*1024*1024*1024; // 18*1024*1024*1024llu; // sizeof(unsigned int)*G.numEdges;
        // restricted_gpu_mem = restricted_gpu_mem / 3;
        const size_t page_size = REQUEST_SIZE;
        const size_t numPages = restricted_gpu_mem/page_size;

        printf("function: %s line: %d\n", __FILE__, __LINE__);
        alloc_global_host_content(host_post, host_poll, keys);
        printf("function: %s line: %d\n", __FILE__, __LINE__);

        ret1 = hipDeviceSynchronize();
        printf("ret: %d\n", ret1);
        if(hipSuccess != ret1){    
            return -1;
        }

        ret1 = hipDeviceSynchronize();
        printf("ret: %d\n", ret1);
        if(hipSuccess != ret1){    
            return -1;
        }
        
        printf("restricted_gpu_mem: %zu\n", restricted_gpu_mem);
        start_page_queue<<<1, 1>>>(/*s_ctx->gpu_buf_size*/restricted_gpu_mem, page_size);
        ret1 = hipDeviceSynchronize();
        printf("ret: %d\n", ret1);
        if(hipSuccess != ret1){    
            return -1;
        }
    }

    DATA_TYPE* a;
	DATA_TYPE* x1;
	DATA_TYPE* x2;
    DATA_TYPE* x1_uvm;
	DATA_TYPE* x2_uvm;
	DATA_TYPE* y_1;
	DATA_TYPE* y_2;
    size_t size = N*N*sizeof(DATA_TYPE);

    a = (DATA_TYPE*)malloc(size);
	x1 = (DATA_TYPE*)malloc(N*sizeof(DATA_TYPE));
	x2 = (DATA_TYPE*)malloc(N*sizeof(DATA_TYPE));
    x1_uvm = (DATA_TYPE*)malloc(N*sizeof(DATA_TYPE));
	x2_uvm = (DATA_TYPE*)malloc(N*sizeof(DATA_TYPE));
	y_1 = (DATA_TYPE*)malloc(N*sizeof(DATA_TYPE));
	y_2 = (DATA_TYPE*)malloc(N*sizeof(DATA_TYPE));

    printf("Allocation finished on GPU. Initializing...\n"); 
    init_array(a, x1, x2, y_1, y_2);
    memcpy(x1_uvm, x1, N*sizeof(DATA_TYPE));
    memcpy(x2_uvm, x2, N*sizeof(DATA_TYPE));

    unsigned long long *direct_new_repr;
    if(rdma_flag){
        printf("Allocation finished. Calling mvtRDMA\n");
        mvtCuda_rdma(a, x1, x2, y_1, y_2);
        hipFree(s_ctx->gpu_buffer);
    }

    printf("Allocation finished. Calling mvtCUDA\n");
    mvtCuda(a, x1_uvm, x2_uvm, y_1, y_2);

    compareResults(x1, x1_uvm, x2, x2_uvm);

    printf("oversubs ratio: %d\n", oversubs_ratio_macro-1);
    
	return 0;
}

__global__ void kernel_coalesce_new_repr_rdma(bool *curr_visit, size_t n, bool *next_visit, uint64_t new_size, unsigned int *new_vertex_list,
                                unsigned int *new_offset, rdma_buf<unsigned int> *edgeList, unsigned long long *comp, bool *changed) {
    size_t warp_size = 32;
    // Page size in elements (64KB / 4 bytes per unsigned int)
    const size_t pageSize = 4*1024 / sizeof(unsigned int);
    // Elements per warp
    const size_t elementsPerWarp = pageSize / warp_size; // warpSize;
    // Global thread ID
    size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Warp ID within the block
    size_t warpId = tid / warp_size; // warpSize;

    // Thread lane within the warp
    size_t lane = threadIdx.x % warp_size; // warpSize;

    // Determine which page this warp will process
    size_t pageStart = warpId * pageSize;

    // Ensure we don't process out-of-bounds pages
    if (pageStart < n * pageSize) {
        // Process elements within the page
        for (size_t i = 0; i < elementsPerWarp; ++i) {
            size_t elementIdx = pageStart + lane + i * warp_size; // warpSize;
            if (elementIdx < new_size && curr_visit[new_vertex_list[elementIdx]] == true) {
                unsigned int start_vertex = new_vertex_list[elementIdx];
                
                // const uint64_t start = new_offset[elementIdx];
                // const uint64_t shift_start = start & MEM_ALIGN;
                // const uint64_t end = new_offset[elementIdx+1];

                // Process adjacent nodes
                for(size_t j = new_offset[elementIdx]/*&MEM_ALIGN*/; j < new_offset[elementIdx+1]; j += 1) {
                    // if(j >= new_offset[elementIdx]){
                        uint next = (*edgeList)[j];
                        if (comp[next] != comp[start_vertex]) {
                            if (comp[start_vertex] < comp[next]) {
                                // next_target = next;
                                // comp_target = comp[elementIdx];
                                atomicMin(&comp[next], comp[start_vertex]);
                                next_visit[next] = true;
                            }
                            else {
                                // next_target = elementIdx;
                                // comp_target = comp[next];
                                atomicMin(&comp[start_vertex], comp[next]);
                                next_visit[start_vertex] = true;   
                            }
                            *changed = true;
                        }
                    // }
                }
            }
        }
    }
}

__global__ void kernel_coalesce_new_repr_uvm(bool *curr_visit, size_t n, bool *next_visit, uint64_t new_size, unsigned int *new_vertex_list,
                                unsigned int *new_offset, unsigned int *edgeList, unsigned long long *comp, bool *changed) {
    size_t warp_size = 32;
    // Page size in elements (64KB / 4 bytes per unsigned int)
    const size_t pageSize = 4*1024 / sizeof(unsigned int);
    // Elements per warp
    const size_t elementsPerWarp = pageSize / warp_size; // warpSize;
    // Global thread ID
    size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Warp ID within the block
    size_t warpId = tid / warp_size; // warpSize;

    // Thread lane within the warp
    size_t lane = threadIdx.x % warp_size; // warpSize;

    // Determine which page this warp will process
    size_t pageStart = warpId * pageSize;

    // Ensure we don't process out-of-bounds pages
    if (pageStart < n * pageSize) {
        // Process elements within the page
        for (size_t i = 0; i < elementsPerWarp; ++i) {
            size_t elementIdx = pageStart + lane + i * warp_size; // warpSize;
            if (elementIdx < new_size && curr_visit[new_vertex_list[elementIdx]] == true) {
                unsigned int start_vertex = new_vertex_list[elementIdx];
                
                // const uint64_t start = new_offset[elementIdx];
                // const uint64_t shift_start = start & MEM_ALIGN;
                // const uint64_t end = new_offset[elementIdx+1];

                // Process adjacent nodes
                for(size_t j = new_offset[elementIdx]/*&MEM_ALIGN*/; j < new_offset[elementIdx+1]; j += 1) {
                    // if(j >= new_offset[elementIdx]){
                        uint next = edgeList[j];
                        if (comp[next] != comp[start_vertex]) {
                            if (comp[start_vertex] < comp[next]) {
                                // next_target = next;
                                // comp_target = comp[elementIdx];
                                atomicMin(&comp[next], comp[start_vertex]);
                                next_visit[next] = true;
                            }
                            else {
                                // next_target = elementIdx;
                                // comp_target = comp[next];
                                atomicMin(&comp[start_vertex], comp[next]);
                                next_visit[start_vertex] = true;   
                            }
                            *changed = true;
                        }
                    // }
                }
            }
        }
    }
}