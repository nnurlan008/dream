#include <stdio.h>
#include <hip/hip_runtime.h>

#include <unistd.h>

int init_gpu(int gpu){
    int cuda_device_id = 0;
	int cuda_pci_bus_id;
	int cuda_pci_device_id;
	int index;
	hipDevice_t cu_device;
	hipDevice_t cuDevice_selected;

	printf("initializing CUDA...\n");
	hipError_t error = hipInit(gpu);
	if (error != hipSuccess) {
		printf("cuInit(0) returned %d\n", error);
		return -1;
	}
	int deviceCount = 0;
	error = hipGetDeviceCount(&deviceCount);
	if (error != hipSuccess) {
		printf("cuDeviceGetCount() returned %d\n", error);
		return -1;
	}
	/* This function call returns 0 if there are no CUDA capable devices. */
	if (deviceCount == 0) {
		printf("There are no available device(s) that support CUDA\n");
		return -1;
	}
  
	for (int index = 0; index < deviceCount; index++) {
		if(hipDeviceGet(&cu_device, index) != hipSuccess) return -1;
		hipDeviceGetAttribute(&cuda_pci_bus_id, hipDeviceAttributePciBusId , cu_device);
		hipDeviceGetAttribute(&cuda_pci_device_id, hipDeviceAttributePciDeviceId , cu_device);
		printf("CUDA device %d: PCIe address is %02X:%02X\n", index, (unsigned int)cuda_pci_bus_id, (unsigned int)cuda_pci_device_id);
	}


	if(hipDeviceGet(&cuDevice_selected, cuda_device_id) != hipSuccess){
		printf("error on cuDeviceGet\n");
		return -1;
	}
	char name[128];
	if(hipDeviceGetName(name, sizeof(name), cuda_device_id) != hipSuccess){
		printf("error on cuDeviceGetName\n");
		return -1;
	}
	printf("[pid = %d, dev = %d] device name = [%s]\n", getpid(), cuDevice_selected, name);

    size_t free_memory, total_memory;
    if(hipSuccess != hipMemGetInfo(&free_memory, &total_memory)){
        printf("error on cudaMemGetInfo\n");
        return -1;
    }
    printf("free memory: %zu, total_memory: %zu\n", free_memory/(1024 * 1024), total_memory/(1024 * 1024));
    return 0;
}


__global__
void initWith(float num, float *a, int N)
{

    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    
        
    for(int i = index; i < N; i += stride)
    {
        a[i] = num;
    }
}

__global__
void add_vectors_UVM(float *result, float *a, float *b, int N)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    
    for(int i = index; i < N; i += stride)
    {
        result[i] = a[i] + b[i];
    }
}

void checkElementsAre(float target, float *vector, int N)
{
    for(int i = 0; i < N; i++)
    {
        if(vector[i] != target)
        {
            printf("FAIL: vector[%d] - %0.0f does not equal %0.0f\n", i, vector[i], target);
            exit(1);
        }
    }
    printf("Success! All values calculated correctly.\n");
}

int main()
{
    int deviceId;
    int numberOfSMs;
    init_gpu(0);
    hipGetDevice(&deviceId);
    hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);

    printf("sizeof(float): %d\n", sizeof(float));
    const int N = 256*1024*1024/sizeof(float); // (2<<24)/8;
    size_t size = N * sizeof(float);
    printf("size: %d MB\n", size/1024/1024);
    float *a;
    float *b;
    float *c;

    hipMallocManaged(&a, size);
    hipMallocManaged(&b, size);
    hipMallocManaged(&c, size);

    // cudaMemPrefetchAsync(a, size, deviceId);
    // cudaMemPrefetchAsync(b, size, deviceId);
    // cudaMemPrefetchAsync(c, size, deviceId);

    size_t threadsPerBlock;
    size_t numberOfBlocks;

    threadsPerBlock = 256;
    numberOfBlocks = 2048; // 32 * numberOfSMs;

    hipError_t addVectorsErr;
    hipError_t asyncErr;

    for (int i = 0; i < N; i++){
        a[i] = 2;
        b[i] = 2;
    }

    // initWith<<<numberOfBlocks, threadsPerBlock>>>(3, a, N);
    // initWith<<<numberOfBlocks, threadsPerBlock>>>(4, b, N);
    // initWith<<<numberOfBlocks, threadsPerBlock>>>(0, c, N);
    hipEvent_t event1, event2;
    hipEventCreate(&event1);
    hipEventCreate(&event2);


    // clock_gettime(CLOCK_REALTIME, &start);
    hipEventRecord(event1, (hipStream_t)0); //where 0 is the default stream

    add_vectors_UVM<<<numberOfBlocks, threadsPerBlock>>>(c, a, b, N);
    hipEventRecord(event2, (hipStream_t) 0);
    
    addVectorsErr = hipGetLastError();
    if(addVectorsErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(addVectorsErr));

    asyncErr = hipDeviceSynchronize();
    if(asyncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(asyncErr));

    hipEventSynchronize(event1); //optional
    hipEventSynchronize(event2); //wait for the event to be executed!

    //calculate time
    float dt_ms;
    hipEventElapsedTime(&dt_ms, event1, event2);
    printf("dt_ms: %f\n", dt_ms);
    // cudaMemPrefetchAsync(c, size, cudaCpuDeviceId);

    checkElementsAre(4, c, N);

    hipFree(a);
    hipFree(b);
    hipFree(c);
}