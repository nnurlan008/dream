#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <string>
#include <cstring>
#include <math.h>
#include <chrono>
#include <stdlib.h>

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

using namespace std;
// using namespace std;


// extern "C"{
//   #include "rdma_utils.h"
// } 

// #include "../../src/rdma_utils.cuh"
#include <time.h>
// #include "../../include/runtime_prefetching.h"
// #include "../../include/runtime_eviction.h"
#include "../../include/runtime_prefetching_2nic.h"

// #include "../../include/runtime_micro.h"

#define MAX_TRIPS 1000000000llu  // Set a maximum number of trips
#define LINE_LENGTH 256   // Set the maximum line length

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.05

//define a small float value
#define SMALL_FLOAT_VAL 0.00000001f

/* Problem size. */
#define NX 4096*16llu
#define NY 4096*16llu

#define BLOCK_NUM 1024ULL
#define MYINFINITY 2147483647llu

#define BLOCK_SIZE 1024
#define WARP_SHIFT 5
#define WARP_SIZE 32

#define THRESHOLD_SECONDS 9000

typedef float DATA_TYPE;

/* Thread block dimensions */
#define DIM_THREAD_BLOCK_X 1024
#define DIM_THREAD_BLOCK_Y 1

#define GPU 0

#ifndef M_PI
#define M_PI 3.14159
#endif

__device__ rdma_buf<unsigned int> D_adjacencyList;

__global__ void test(rdma_buf<unsigned int> *a/*, rdma_buf<int> *b, rdma_buf<int> *c*/);


// Kernel
__global__ void add_vectors_uvm(int *a, int *b, int *c, int size)
{
	int id = blockDim.x * blockIdx.x + threadIdx.x;
	// if(id < size) {
		c[id] = a[id] + b[id];
		// printf("c[%d]: %d\n", id, c[id]);
	// }
}

#define htonl(x)  ((((uint32_t)(x) & 0xff000000) >> 24) |\
                   (((uint32_t)(x) & 0x00ff0000) >>  8) |\
                   (((uint32_t)(x) & 0x0000ff00) <<  8) |\
                   (((uint32_t)(x) & 0x000000ff) << 24))

#define WARP_SIZE 32

void delay(int number_of_seconds)
{
    // Converting time into milli_seconds
    int milli_seconds = 1000000 * number_of_seconds;
 
    // Storing start time
    clock_t start_time = clock();
 
    // looping till required time is not achieved
    while (clock() < start_time + milli_seconds)
        ;
}

enum { NS_PER_SECOND = 1000000000 };

void sub_timespec(struct timespec t1, struct timespec t2, struct timespec *td)
{
    td->tv_nsec = t2.tv_nsec - t1.tv_nsec;
    td->tv_sec  = t2.tv_sec - t1.tv_sec;
    if (td->tv_sec > 0 && td->tv_nsec < 0)
    {
        td->tv_nsec += NS_PER_SECOND;
        td->tv_sec--;
    }
    else if (td->tv_sec < 0 && td->tv_nsec > 0)
    {
        td->tv_nsec -= NS_PER_SECOND;
        td->tv_sec++;
    }
}

__device__ float AtomicAdd(float *address, float value) {
    // Convert address to integer representation (since atomicCAS works on integers)
    uint32_t *address_as_int = (uint32_t *)address;
    uint32_t old = *address_as_int, assumed;

    // Loop to perform atomic addition
    do {
        assumed = old;
        // Convert the integer bits back to a float and perform the addition
        float old_f = __int_as_float(assumed);
        float new_f = old_f + value;

        // Use atomicCAS to try and set the new value (converted to int)
        old = atomicCAS(address_as_int, assumed, __float_as_int(new_f));
    } while (assumed != old);  // Retry if the value changed during the process

    // Return the old value (before the addition)
    return __int_as_float(old);
}


void usage(const char *argv0)
{
  fprintf(stderr, "usage: %s <mode> <server-address> <server-port>\n  mode = \"read\", \"write\"\n", argv0);
  exit(1);
}


#define check_cuda_error(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }

}

__global__ void transfer(size_t size, rdma_buf<DATA_TYPE> *d_adjacencyList)
{
    size_t id = blockDim.x * blockIdx.x + threadIdx.x;
    size_t stride = blockDim.x * gridDim.x;
    
        for (size_t i = id; i < size ; i += stride)
        {
            DATA_TYPE y = (*d_adjacencyList)[i];
        }
}

__global__ void check(size_t size, rdma_buf<DATA_TYPE> *d_adjacencyList, DATA_TYPE *a)
{
    size_t id = blockDim.x * blockIdx.x + threadIdx.x;
    size_t stride = blockDim.x * gridDim.x;
    
        for (size_t i = id; i < size ; i += stride)
        {
            DATA_TYPE y = (*d_adjacencyList)[i];
            if(a[i] != y){
                printf("y: %f %f ", y, a[i]);
            }
        }
}

__global__ void assign_array(rdma_buf<unsigned int> *adjacencyList){
    D_adjacencyList = *adjacencyList;
    printf("D_adjacencyList.d_TLB[0].state: %d\n", D_adjacencyList.d_TLB[0].state);
    printf("D_adjacencyList.d_TLB[0].device_address: %p\n", D_adjacencyList.d_TLB[0].device_address);
}

int alloc_global_cont(struct post_content *post_cont, struct poll_content *poll_cont, struct server_content_2nic *post_cont2, 
                      struct gpu_memory_info gpu_mem){
    struct post_content *d_post;
    struct poll_content *d_poll;
    struct server_content_2nic *d_post2;

    hipError_t ret0 = hipMalloc((void **)&d_post, sizeof(struct post_content));
    if(ret0 != hipSuccess){
        printf("Error on allocation post content!\n");
        return -1;
    }
    ret0 = hipMalloc((void **)&d_poll, sizeof(struct poll_content));
    if(ret0 != hipSuccess){
        printf("Error on allocation poll content!\n");
        return -1;
    }
    printf("sizeof(struct post_content): %d, sizeof(struct poll_content): %d\n", sizeof(struct post_content), sizeof(struct poll_content));
    ret0 = hipMemcpy(d_post, post_cont, sizeof(struct post_content), hipMemcpyHostToDevice);
    if(ret0 != hipSuccess){
        printf("Error on post copy!\n");
        return -1;
    }
    ret0 = hipMemcpy(d_poll, poll_cont, sizeof(struct poll_content), hipMemcpyHostToDevice);
    if(ret0 != hipSuccess){
        printf("Error on poll copy!\n");
        return -1;
    }

    ret0 = hipMalloc((void **)&d_post2, sizeof(struct server_content_2nic));
    if(ret0 != hipSuccess){
        printf("Error on allocation post content!\n");
        return -1;
    }
    ret0 = hipMemcpy(d_post2, post_cont2, sizeof(struct server_content_2nic), hipMemcpyHostToDevice);
    if(ret0 != hipSuccess){
        printf("Error on poll copy!\n");
        return -1;
    }

    // hipSetDevice(0);
    alloc_content<<<1,1>>>(d_post, d_poll);
    alloc_global_content<<<1,1>>>(d_post, d_poll, d_post2, gpu_mem);
    ret0 = hipDeviceSynchronize();
    if(ret0 != hipSuccess){
        printf("Error on alloc_content!\n");
        return -1;
    }
    return 0;
}


__device__ size_t sum_page_faults = 0;

__global__ void
print_retires(void){
    // size_t max = cq_wait[0];
    // for (size_t i = 0; i < 128; i++)
    // {
    //     if(max < cq_wait[i]) max = cq_wait[i];
    // }
    sum_page_faults += g_qp_index;
    printf("g_qp_index: %llu sum page fault: %llu\n", g_qp_index, sum_page_faults);
    g_qp_index = 0;
    // for (size_t i = 0; i < 128; i++)
    // {
    //     max = 0;
    // }
}

__global__ // __launch_bounds__(1024,2) 
void
calculate_opt(size_t n, size_t size, rdma_buf<unsigned int> *rdma_array, unsigned int *array) {

    // Page size in elements (64KB / 4 bytes per unsigned int)
    const size_t pageSize = 8*1024 / sizeof(unsigned int);
    // Elements per warp
    const size_t elementsPerWarp = pageSize / warpSize;

    // Global thread ID
    size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    // if(tid == 0) printf("warpSize: %d\n", warpSize);
    // Warp ID within the block
    size_t warpId = tid / warpSize;

    // Thread lane within the warp
    size_t lane = threadIdx.x % warpSize;

    // Determine which page this warp will process
    size_t pageStart = warpId * pageSize;

    // Ensure we don't process out-of-bounds pages
    if (pageStart < n * pageSize) {
        
        // Process elements within the page
        // for (size_t i = 0; i < elementsPerWarp; ++i) {
        //     size_t elementIdx = pageStart + lane + i * warpSize;
            uint end = (warpId + 1)*pageSize > size ? size : (warpId + 1)*pageSize;
            for(size_t j = warpId*pageSize + lane; j < end; j += warpSize) {
                uint end_edge = (*rdma_array)[j]; // shared_data[j - pageStart];
                array[j] = end_edge;
            }
        // }
    }
}


__global__ void transfer_opt(size_t n, rdma_buf<unsigned int> *rdma_array) {


    // Page size in elements (64KB / 4 bytes per unsigned int)
    const size_t pageSize = 4*1024 / sizeof(unsigned int);
    // Elements per warp
    const size_t elementsPerWarp = pageSize / warpSize;

    // Global thread ID
    size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    // if(tid == 0) printf("warpSize: %d\n", warpSize);
    // Warp ID within the block
    size_t warpId = tid / warpSize;

    // Thread lane within the warp
    size_t lane = threadIdx.x % warpSize;

    // Determine which page this warp will process
    size_t pageStart = warpId * pageSize;

    // Ensure we don't process out-of-bounds pages
    if (pageStart < n * pageSize) {
        
        // Process elements within the page
        // for (size_t i = 0; i < elementsPerWarp; ++i) {
        //     size_t elementIdx = pageStart + lane + i * warpSize;
                
            for(size_t j = warpId*pageSize + lane; j < (warpId + 1)*pageSize; j += warpSize) {
                uint end_edge = (*rdma_array)[j]; // shared_data[j - pageStart];
            }
        // }
    }
}

void compute_benchmark(){

    hipError_t ret;
    unsigned int *cuda_array, *h_cuda_array;
    uint64_t numblocks_update, numthreads, numblocks_kernel;
    double avg_milliseconds;
    float milliseconds;
    size_t num_elements, size = 12*1024*1024*1024llu; 
    num_elements = size/sizeof(uint);
    hipEvent_t start, end;

    rdma_buf<unsigned int> *rdma_array;
    check_cuda_error(hipMallocManaged((void **) &rdma_array, sizeof(rdma_buf<unsigned int>)));
    rdma_array->start(num_elements *sizeof(unsigned int), GPU, NULL);
    for(size_t i = 0; i < num_elements; i++){
        rdma_array->local_buffer[i] = 14;
    }

    check_cuda_error(hipEventCreate(&start));
    check_cuda_error(hipEventCreate(&end));

    h_cuda_array = new uint[num_elements];
    check_cuda_error(hipMalloc((void **) &cuda_array, size));

    // numblocks_update = ((numVertex + numthreads) / numthreads);
    dim3 blockDim_kernel(numthreads, (numblocks_kernel+numthreads)/numthreads);
    // dim3 blockDim_kernel(BLOCK_SIZE, (numblocks_kernel+BLOCK_SIZE)/BLOCK_SIZE);
    numthreads = BLOCK_SIZE;
    // numblocks_update = ((numVertex + numthreads) / numthreads);
    dim3 blockDim_update(BLOCK_SIZE, (numblocks_update+BLOCK_SIZE)/BLOCK_SIZE);

    avg_milliseconds = 0.0f;

    ret = hipDeviceSynchronize();
    printf("ret: %d hipGetLastError: %d\n", ret, hipGetLastError());
    auto start_chrono = std::chrono::steady_clock::now();
    printf("starting kernel\n");
    size_t n_pages = size/(8*1024);

    check_cuda_error(hipEventRecord(start, (hipStream_t) 1));
    numthreads = 1024;
    calculate_opt<<<(n_pages*32)/numthreads+1, numthreads>>>(num_elements, num_elements, rdma_array, cuda_array);
    check_cuda_error(hipEventRecord(end, (hipStream_t) 1));

    ret = hipDeviceSynchronize();               
    auto end_chrono = std::chrono::steady_clock::now();
    long duration = std::chrono::duration_cast<std::chrono::milliseconds>(end_chrono - start_chrono).count();
    print_retires<<<1,1>>>();
    printf("ret: %d hipGetLastError: %d\n", ret, hipGetLastError());
    printf("Elapsed time in milliseconds : %li ms\n\n", duration);
    // printf("ret: %d hipGetLastError: %d\n", ret, hipGetLastError());
    
    check_cuda_error(hipMemcpy(h_cuda_array, cuda_array, size, hipMemcpyDeviceToHost));
    size_t num_errs = 0;
    for (size_t i = 0; i < num_elements; i++)
    {
        if(h_cuda_array[i] != 14){
            num_errs++;
            // printf("i: %d h_cuda_array[i]: %d\n", i, h_cuda_array[i]);
        }
    }
    

    
    check_cuda_error(hipEventSynchronize(start));
    check_cuda_error(hipEventSynchronize(end));
    check_cuda_error(hipEventElapsedTime(&milliseconds, start, end));
    printf("CUDA elapsed time in milliseconds : %0.3f ms num_errs: %llu bw: %.2f REQUEST_SIZE: %d\n\n", milliseconds, num_errs, (float) size/(milliseconds*0.001*1024*1024*1024), REQUEST_SIZE/1024);

}

void transfer_benchmark(){
    hipError_t ret;
    
    uint64_t numblocks_update, numthreads, numblocks_kernel;
    double avg_milliseconds;
    float milliseconds;
    size_t num_elements, size = 1*1024*1024*1024llu; 
    num_elements = size/sizeof(uint);
    
    hipEvent_t start, end;

    rdma_buf<unsigned int> *rdma_array;
    check_cuda_error(hipMallocManaged((void **) &rdma_array, sizeof(rdma_buf<unsigned int>)));

    rdma_array->start(num_elements *sizeof(unsigned int), GPU, NULL);

    for(size_t i = 0; i < num_elements; i++){
        rdma_array->local_buffer[i] = 14;
    }

   
    check_cuda_error(hipEventCreate(&start));
    check_cuda_error(hipEventCreate(&end));
   

    // numblocks_update = ((numVertex + numthreads) / numthreads);

    dim3 blockDim_kernel(numthreads, (numblocks_kernel+numthreads)/numthreads);
    // dim3 blockDim_kernel(BLOCK_SIZE, (numblocks_kernel+BLOCK_SIZE)/BLOCK_SIZE);
    numthreads = BLOCK_SIZE;
    // numblocks_update = ((numVertex + numthreads) / numthreads);
    dim3 blockDim_update(BLOCK_SIZE, (numblocks_update+BLOCK_SIZE)/BLOCK_SIZE);

    avg_milliseconds = 0.0f;

    printf("Initialization done\n");
    fflush(stdout);

    
    ret = hipDeviceSynchronize();
    printf("ret: %d hipGetLastError: %d\n", ret, hipGetLastError());
    

    auto start_chrono = std::chrono::steady_clock::now();
    
    printf("starting kernel\n");
    size_t n_pages = size/(4*1024);

    check_cuda_error(hipEventRecord(start, (hipStream_t) 1));
    transfer_opt<<<(n_pages*32)/512+1, 512>>>(num_elements, rdma_array);
    check_cuda_error(hipEventRecord(end, (hipStream_t) 1));

    ret = hipDeviceSynchronize();               
    auto end_chrono = std::chrono::steady_clock::now();
    long duration = std::chrono::duration_cast<std::chrono::milliseconds>(end_chrono - start_chrono).count();
    print_retires<<<1,1>>>();
    printf("ret: %d hipGetLastError: %d\n", ret, hipGetLastError());
    printf("Elapsed time in milliseconds : %li ms\n\n", duration);
    // printf("ret: %d hipGetLastError: %d\n", ret, hipGetLastError());
    

    
    check_cuda_error(hipEventSynchronize(start));
    check_cuda_error(hipEventSynchronize(end));
    check_cuda_error(hipEventElapsedTime(&milliseconds, start, end));
    printf("CUDA elapsed time in milliseconds : %li ms\n\n", milliseconds);

}

// Custom function to convert string to integer
__device__ int my_atoi(const char *str) {
    int result = 0;
    int sign = 1;
    int i = 0;

    // Handle negative numbers
    if (str[0] == '-') {
        sign = -1;
        i++;
    }

    for (; str[i] != '\0' && str[i] != '\n'; i++) {
        result = result * 10 + (str[i] - '0');
    }
    return result * sign;
}

// Custom function to convert string to float
__forceinline__
__device__ float my_atof(const char *str) {
    float result = 0.0f;
    float divisor = 1.0f;
    int i = 0;
    int sign = 1;

    // Handle negative numbers
    if (str[0] == '-') {
        sign = -1;
        i++;
    }

    // Convert integer part
    for (; str[i] != '\0' && str[i] != '.' && str[i] != '\n'; i++) {
        result = result * 10.0f + (str[i] - '0');
    }

    // Convert decimal part
    if (str[i] == '.') {
        i++;
        for (; str[i] != '\0' && str[i] != '\n'; i++) {
            result = result * 10.0f + (str[i] - '0');
            divisor *= 10.0f;  // Increment divisor for each decimal place
        }
    }

    return sign * (result / divisor);
}

__forceinline__
__device__ void parse_trip(const char *line, float &trip_seconds, float &trip_miles, float &fare_amount, float &tip_amount, float &tolls, float &extra) {
    // Manually parse the line
    char buffer[LINE_LENGTH];
    int index = 0;
    int field = 0;
    
    for (int i = 0; line[i] != '\0' && i < LINE_LENGTH; i++) {
        if (line[i] == ',' || line[i] == '\n') {
            buffer[index] = '\0'; // Null-terminate the string
            switch (field) {
                // case 0: // VendorID
                //     *vendorID = my_atoi(buffer);
                //     break;
                // case 6: // Pickup latitude
                //     pickup_latitude = my_atof(buffer);
                //     break;
                // case 5: // Pickup longitude
                //     *pickup_longitude = my_atof(buffer);
                //     break;
                case 4: // Trip Distance
                    trip_seconds = my_atof(buffer);
                    break;
                case 5: // Trip Distance
                    trip_miles = my_atof(buffer);
                    break;
                case 10: // fare_amount
                    fare_amount = my_atof(buffer);
                    break;
                case 11: // tip_amount
                    tip_amount = my_atof(buffer);
                    break;
                case 12: // mta_tax
                    tolls = my_atof(buffer);
                    break;
                case 13: // extra
                    extra = my_atof(buffer);
                    break;
            }
            field++;
            index = 0; // Reset index for next field
        } else {
            buffer[index++] = line[i]; // Collect character into buffer
        }
    }
}

__global__ __launch_bounds__(1024,2)
void process_trips(const char *buffer, int count, float *total_amount, float *total_miles) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < count) {
        
        // Declare an array to hold the line
        char line[LINE_LENGTH];  // Create a character array for one line

        // Copy the line from the buffer into the line array
        for (int i = 0; i < LINE_LENGTH; ++i) {
            line[i] = buffer[idx * LINE_LENGTH + i];
        }

        int vendorID;
        float trip_seconds;
        float trip_miles;
        float fare_amount;
        float tip_amount;
        float tolls;
        float extra;
        
        // Parse the trip details
        if(idx == 0 || idx == 1 || idx == 2) printf("first line: %s\n", line);
        parse_trip(line, trip_seconds, trip_miles, fare_amount, tip_amount, tolls, extra);

        if(trip_seconds > 1000 && trip_miles < 1050) {
            float local_total = fare_amount - extra - tolls + tip_amount;
            // size_t value = (size_t) local_total;
            // printf("local_total : %llu\n", value);
            // *total_amount += 1;
            // if(trip_distance < 1000){
                // printf("trip_seconds: %f trip_miles: %f, fare_amount: %f, extra: %f tolls: %f tip_amount: %f\nline: %s\n", 
                //         trip_seconds, trip_miles, fare_amount, extra, tolls, tip_amount, line);
            //     // printf("line: %s\n", line);
                AtomicAdd(total_amount, local_total);
                AtomicAdd(total_miles, trip_miles);
            // }
        }

        // Example operation: Store or print the parsed values (printing from device is not recommended)
        // printf("Vendor ID: %d, Pickup Latitude: %f, Pickup Longitude: %f\n", vendorID, pickup_latitude, pickup_longitude);
    }
}

__global__ __launch_bounds__(1024,2)
void process_trips_rdma(rdma_buf<char> *buffer, int count, float *total_amount, float *total_miles) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < count) {
        
        // Declare an array to hold the line
        char line[LINE_LENGTH];  // Create a character array for one line

        // Copy the line from the buffer into the line array
        for (int i = 0; i < LINE_LENGTH; ++i) {
            line[i] = (*buffer)[idx * LINE_LENGTH + i];
        }

        int vendorID;
        float trip_seconds;
        float trip_miles;
        float fare_amount;
        float tip_amount;
        float tolls;
        float extra;
        
        // Parse the trip details
        // if(idx == 0 || idx == 1 || idx == 2) printf("first line: %s\n", line);
        parse_trip(line, trip_seconds, trip_miles, fare_amount, tip_amount, tolls, extra);

        if(trip_seconds > 1000 && trip_miles < 1050) {
            float local_total = fare_amount - extra - tolls + tip_amount;
            // size_t value = (size_t) local_total;
            // printf("local_total : %llu\n", value);
            // *total_amount += 1;
            // if(trip_distance < 1000){
                // printf("trip_seconds: %f trip_miles: %f, fare_amount: %f, extra: %f tolls: %f tip_amount: %f\nline: %s\n", 
                //         trip_seconds, trip_miles, fare_amount, extra, tolls, tip_amount, line);
            //     // printf("line: %s\n", line);
                AtomicAdd(total_amount, local_total);
                AtomicAdd(total_miles, trip_miles);
            // }
        }

        // Example operation: Store or print the parsed values (printing from device is not recommended)
        // printf("Vendor ID: %d, Pickup Latitude: %f, Pickup Longitude: %f\n", vendorID, pickup_latitude, pickup_longitude);
    }
}

void trips_seconds(char *file){

}

// Function to read CSV data into a single buffer
int read_csv(const char *filename, char *&buffer, size_t *count) {
    FILE *file = fopen(filename, "r");
    if (!file) {
        perror("Failed to open file read_csv");
        return -1;
    }

    fseek(file, 0L, SEEK_END);
    size_t file_size = ftell(file);
    fseek(file, 0L, SEEK_SET);  // Reset to beginning of the file

    // Adjust size based on file size if needed
    size_t max_size = (file_size > (MAX_TRIPS * LINE_LENGTH)) ? file_size : (MAX_TRIPS * LINE_LENGTH);
    printf("max_size: %llu\n", max_size);


    size_t size = MAX_TRIPS * LINE_LENGTH * sizeof(char);
    printf("size: %llu\n", size);
    // buffer = (char *)malloc(size);
    // check_cuda_error(hipHostMalloc(&buffer, size));
    if (!buffer) {
        perror("Failed to allocate memory");
        fclose(file);
        return -1;
    }
    printf("size1: %llu\n", size);
    *count = 0;
    while (fgets(buffer + (*count * LINE_LENGTH), LINE_LENGTH, file) && *count < MAX_TRIPS) {
        (*count)++;
    }
    printf("size2: %llu\n", size);
    fclose(file);
    return 0;
}

int rapids_CUDA(char *file){
    hipEvent_t event1, event2;
    hipEventCreate(&event1);
    hipEventCreate(&event2);
    hipError_t ret;
    char *buffer, *d_buffer;
    size_t count;
    float *total_amount, *h_total, h_miles, *d_miles;
    size_t size = MAX_TRIPS * LINE_LENGTH * sizeof(char);
    h_total = (float *) malloc(sizeof(float));
    buffer = (char *)malloc(size);
    // hipError_t err = hipHostMalloc((void**)&buffer, size);
    printf("line: %d\n", __LINE__);
    // buffer[0] = 12;
    // if (err != hipSuccess) {
    //     printf("hipHostMalloc failed: %s\n", hipGetErrorString(err));
    //     return -1;
    // }
    // Step 1: Read CSV into a single buffer
    if (read_csv(file, buffer, &count) != 0) {
        return -1;
    }

    printf("count: %llu\n", count);

    hipDeviceSynchronize();
    auto start = std::chrono::steady_clock::now();                
    hipEventRecord(event1, (hipStream_t)1);

    // Step 2: Allocate device memory
    check_cuda_error(hipMallocManaged((void **)&d_buffer, count * LINE_LENGTH * sizeof(char)));
    memcpy(d_buffer, buffer, count * LINE_LENGTH * sizeof(char));
    check_cuda_error(hipMalloc((void **)&total_amount, sizeof(float)));
    check_cuda_error(hipMalloc((void **)&d_miles, sizeof(float)));
    check_cuda_error(hipMemset(total_amount, 0, sizeof(float)));

    // Step 3: Copy data to device
    // check_cuda_error(hipMemcpy(d_buffer, buffer, count * LINE_LENGTH * sizeof(char), hipMemcpyHostToDevice));

    // Step 4: Launch kernel
    int threadsPerBlock = 1024;
    int blocksPerGrid = (count + threadsPerBlock - 1) / threadsPerBlock;
    ret = hipDeviceSynchronize();
    printf("ret: %d hipGetLastError(): %d\n", ret, hipGetLastError());
    process_trips<<<blocksPerGrid, threadsPerBlock>>>(d_buffer, count, total_amount, d_miles);
    ret = hipDeviceSynchronize();
    printf("ret: %d hipGetLastError(): %d\n", ret, hipGetLastError());

    check_cuda_error(hipMemcpy(h_total, total_amount, sizeof(float), hipMemcpyDeviceToHost));
    check_cuda_error(hipMemcpy(&h_miles, d_miles, sizeof(float), hipMemcpyDeviceToHost));
    hipDeviceSynchronize();

    hipEventRecord(event2, (hipStream_t) 1);
            
    hipEventSynchronize(event1); //optional
    hipEventSynchronize(event2); //wait for the event to be executed!

    auto end = std::chrono::steady_clock::now();
    long duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
    printf("Elapsed time for direct transfer  ms : %li ms.\n\n", duration);
    
    float dt_ms;
    hipEventElapsedTime(&dt_ms, event1, event2);
    printf("The execution time with direct transfer on GPU: %f ms\n", dt_ms);

    printf("h_total: %f\n", *h_total);
    printf("h_miles: %f\n", h_miles);

    // Step 5: Free memory
    free(buffer);
    free(h_total);
    hipFree(d_buffer);
    hipFree(total_amount);

    return 0;
}

int rapids_RDMA(char *filename){

    rdma_buf<char> *rdma_buffer;
    size_t count;
    check_cuda_error(hipMallocManaged((void **)&rdma_buffer, sizeof(rdma_buf<char>)));
    
    // char *buffer;
    FILE *file = fopen(filename, "r");
    if (!file) {
        perror("Failed to open file rapids_RDMA");
        return -1;
    }
    size_t size = MAX_TRIPS * LINE_LENGTH * sizeof(char);
    printf("size: %llu\n", size);
    rdma_buffer->start(size, GPU, NULL);
    printf("size0: %llu\n", size);
    // buffer = (char *) malloc(size);
    // if (!buffer) {
    //     perror("Failed to allocate memory");
    //     fclose(file);
    //     return -1;
    // }
    
    printf("size1: %llu\n", size);

    count = 0;
    while(fgets(rdma_buffer->local_buffer + (count * LINE_LENGTH), LINE_LENGTH, file) && count < MAX_TRIPS) {
        (count)++;
    }
    printf("size2: %llu\n", size);
    fclose(file);
    // return 0;




    hipEvent_t event1, event2;
    hipEventCreate(&event1);
    hipEventCreate(&event2);
    hipError_t ret;
    
    
    float *total_amount, *h_total, *d_miles, h_miles;
    h_total = (float *) malloc(sizeof(float));

    // Step 1: Read CSV into a single buffer
    // if (read_csv(file, buffer, &count) != 0) {
    //     return -1;
    // }

    printf("count: %llu\n", count);

    auto start = std::chrono::steady_clock::now();                
    hipEventRecord(event1, (hipStream_t)1);

    // Step 2: Allocate device memory
    // check_cuda_error(hipMalloc((void **)&d_buffer, count * LINE_LENGTH * sizeof(char)));
    check_cuda_error(hipMalloc((void **)&d_miles, sizeof(float)));
    check_cuda_error(hipMalloc((void **)&total_amount, sizeof(float)));
    check_cuda_error(hipMemset(total_amount, 0, sizeof(float)));

    // Step 3: Copy data to device
    // check_cuda_error(hipMemcpy(d_buffer, buffer, count * LINE_LENGTH * sizeof(char), hipMemcpyHostToDevice));

    // Step 4: Launch kernel
    int threadsPerBlock = 1024;
    int blocksPerGrid = (count + threadsPerBlock - 1) / threadsPerBlock;
    ret = hipDeviceSynchronize();
    printf("ret: %d hipGetLastError(): %d\n", ret, hipGetLastError());
    process_trips_rdma<<<blocksPerGrid, threadsPerBlock>>>(rdma_buffer, count, total_amount, d_miles);
    ret = hipDeviceSynchronize();
    printf("ret: %d hipGetLastError(): %d\n", ret, hipGetLastError());

    check_cuda_error(hipMemcpy(h_total, total_amount, sizeof(float), hipMemcpyDeviceToHost));
    check_cuda_error(hipMemcpy(&h_miles, d_miles, sizeof(float), hipMemcpyDeviceToHost));
    hipDeviceSynchronize();

    hipEventRecord(event2, (hipStream_t) 1);
            
    hipEventSynchronize(event1); //optional
    hipEventSynchronize(event2); //wait for the event to be executed!

    auto end = std::chrono::steady_clock::now();
    long duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
    printf("Elapsed time for rdma transfer  ms : %li ms.\n\n", duration);
    
    float dt_ms;
    hipEventElapsedTime(&dt_ms, event1, event2);
    printf("The cudaEvent execution time with rdma on GPU: %f ms\n", dt_ms);

    printf("h_total: %f\n", *h_total);
    printf("h_miles: %f\n", h_miles);

    // Step 5: Free memory
    // free(buffer);
    // hipFree(d_buffer);

    return 0;
}

// __global__ __launch_bounds__(1024,2)
// void process_trips_uvm_direct(float *uvm_file, int count, float *total_amount, float *total_miles) {
//     size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
//     if (idx < count) {
        
//         // Declare an array to hold the line
//         // char line[LINE_LENGTH];  // Create a character array for one line

//         // int vendorID;
//         float trip_seconds = uvm_file[idx];
        
//         // parse_trip(line, trip_seconds, trip_miles, fare_amount, tip_amount, tolls, extra);
//         // trip_miles < 1050

//         if(trip_seconds > 1000) {

//             // printf("trip_seconds: %f\n", trip_seconds);

//             float trip_miles  = uvm_file[5*idx  + count + 0];
//             float fare_amount = uvm_file[5*idx + count + 1];
//             float tip_amount  = uvm_file[5*idx + count + 2];
//             float tolls       = uvm_file[5*idx + count + 3];
//             float extra       = uvm_file[5*idx + count + 4];

//             float local_total = fare_amount; // - extra - tolls + tip_amount;
            
//                 AtomicAdd(total_amount, local_total);
//                 AtomicAdd(total_miles, trip_miles);
//         }

//         // Example operation: Store or print the parsed values (printing from device is not recommended)
//         // printf("Vendor ID: %d, Pickup Latitude: %f, Pickup Longitude: %f\n", vendorID, pickup_latitude, pickup_longitude);
//     }
// }

__global__ __launch_bounds__(1024,2)
void process_trips_uvm_direct_trip_miles(float *uvm_file, size_t n, size_t count, float *total_amount, float *total_miles, int *array, float *sum) {


    // // Page size in elements (64KB / 4 bytes per unsigned int)
    // const size_t pageSize = REQUEST_SIZE / sizeof(float);
    // // Elements per warp
    // const size_t elementsPerWarp = pageSize / WARP_SIZE;

    // // Global thread ID
    // size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    // // if(tid == 0) printf("warpSize: %d\n", warpSize);
    // // Warp ID within the block
    // size_t warpId = tid / WARP_SIZE;

    // // Thread lane within the warp
    // size_t lane = threadIdx.x % WARP_SIZE; // warpSize;

    // // Determine which page this warp will process
    // size_t pageStart = warpId * pageSize;

    // // Ensure we don't process out-of-bounds pages
    // if (pageStart < n * pageSize) {
        
    //     // Process elements within the page
    //     // for (size_t i = 0; i < elementsPerWarp; ++i) {
    //     //     size_t elementIdx = pageStart + lane + i * warpSize;
    //         size_t end = (warpId + 1)*pageSize > count ? count : (warpId + 1)*pageSize;
    //         for(size_t j = warpId*pageSize + lane; j < end; j += WARP_SIZE) {
    //             float trip_seconds = uvm_file[j];
    //             if(trip_seconds > THRESHOLD_SECONDS) {
    //                 // 
    //                 // if(trip_miles < 3460.0){
    //                     // size_t index = 5*j + count;
    //                     // float trip_miles  = uvm_file[index ];
    //                     // float fare_amount = uvm_file[index + 1];     
    //                     // float extra       = uvm_file[index + 4];
    //                     // float tip_amount  = uvm_file[index + 2];
    //                     // float tolls       = uvm_file[index + 3];

    //                     float trip_miles  = uvm_file[j + count];
    //                     // float fare_amount = uvm_file[j + count*2];
                        
    //                     // float extra       = uvm_file[j + count*5];
    //                     // float tip_amount  = uvm_file[j + count*3];
    //                     // float tolls       = uvm_file[j + count*4];
                        
                        

    //                     // float local_total = fare_amount; // - extra; // + tip_amount; // - tolls;
    //                     // sum[j] += local_total;
    //                     total_miles[j] += trip_miles;
                        
    //                     // atomicAdd(total_amount, local_total);
    //                     // atomicAdd(total_miles, trip_miles);
    //                     // array[j] = 1;
    //                     // atomicAdd(ones, 1);
    //                 // }
    //             }
    //             // else{
    //             //     printf("trip_seconds: %f\n", trip_seconds);
    //             // }
    //         }
    //     // }
    // }


    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < count) {
        float trip_seconds = uvm_file[idx];
        if(trip_seconds > THRESHOLD_SECONDS) {
            float trip_miles  = uvm_file[idx  + count];
            float fare_amount = uvm_file[idx + count*2];
            float extra       = uvm_file[idx + count*5];
            float tip_amount  = uvm_file[idx + count*3];
            float tolls       = uvm_file[idx + count*4];

            float local_total = fare_amount - extra + tip_amount - tolls;
            sum[idx] += local_total;
            total_miles[idx] += trip_miles;
        }
    }
}

__global__ // __launch_bounds__(1024,2)
void process_trips_uvm_direct_trip_miles2(float *uvm_seconds, float *uvm_miles, float *uvm_fare, float *uvm_extra, float *uvm_tips, float *uvm_tolls, 
                    size_t n, size_t count, float *total_amount, float *total_miles, int *array, float *sum) {

    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < count) {
        float trip_seconds = uvm_seconds[idx];
        if(trip_seconds > THRESHOLD_SECONDS) {
            float trip_miles  = uvm_miles[idx];
            float fare_amount = uvm_fare[idx];
            float extra       = uvm_extra[idx];
            float tip_amount  = uvm_tips[idx];
            float tolls       = uvm_tolls[idx];

            float local_total = fare_amount - extra + tip_amount - tolls;
            sum[idx] += local_total;
            total_miles[idx] += trip_miles;
        }
    }
}


int read_bin_uvm(char *filename, size_t &max1, float *&seconds, float *&miles, float *&fare, float *&extra, float *&tips, float *&tolls){
    size_t max = 211670894llu;
    max1 = max;
    float *trip_seconds; // [MAX_TRIPS];
    float *trip_miles;
    float *fare_amount;
    float *tip_amount;
    float *tolls_amount;
    float *extra_amount;
    trip_seconds = (float *) malloc(sizeof(float)*max);
    trip_miles = (float *) malloc(sizeof(float)*max);
    fare_amount = (float *) malloc(sizeof(float)*max);
    tip_amount = (float *) malloc(sizeof(float)*max);
    tolls_amount = (float *) malloc(sizeof(float)*max);
    extra_amount = (float *) malloc(sizeof(float)*max);
    

    size_t res;
    size_t size = max*sizeof(float);
    check_cuda_error(hipMallocManaged((void **)&seconds, size));
    check_cuda_error(hipMallocManaged((void **)&miles, size));
    check_cuda_error(hipMallocManaged((void **)&fare, size));
    check_cuda_error(hipMallocManaged((void **)&extra, size));
    check_cuda_error(hipMallocManaged((void **)&tips, size));
    check_cuda_error(hipMallocManaged((void **)&tolls, size));

    // check_cuda_error(hipMemAdvise(seconds, size, hipMemAdviseSetReadMostly, 0));
    // check_cuda_error(hipMemAdvise(miles, size, hipMemAdviseSetReadMostly, 0));
    // check_cuda_error(hipMemAdvise(fare, size, hipMemAdviseSetReadMostly, 0));
    // check_cuda_error(hipMemAdvise(extra, size, hipMemAdviseSetReadMostly, 0));
    // check_cuda_error(hipMemAdvise(tips, size, hipMemAdviseSetReadMostly, 0));
    // check_cuda_error(hipMemAdvise(tolls, size, hipMemAdviseSetReadMostly, 0));

    auto start = std::chrono::steady_clock::now();
    
    FILE *file = fopen(filename, "rb");
    if (file == NULL) {
        perror("Failed to open file for reading");
        return EXIT_FAILURE;
    }

    res = fread(trip_seconds, sizeof(float), max, file);
    if (res != max) {
        perror("Failed to read all data in trip_seconds");
    }

    res = fread(trip_miles, sizeof(float), max, file);
    if (res != max) {
        perror("Failed to read all data in trip_miles");
    }
    
    res = fread(fare_amount, sizeof(float), max, file);
    if (res != max) {
        perror("Failed to read all data in fare_amount");
    }
    
    res = fread(tip_amount, sizeof(float), max, file);
    if (res != max) {
        perror("Failed to read all data in tip_amount");
    }
    
    res = fread(tolls_amount, sizeof(float), max, file);
    if (res != max) {
        perror("Failed to read all data in tolls");
    }
    
    res = fread(extra_amount, sizeof(float), max, file);
    if (res != max) {
        perror("Failed to read all data in extra");
    }

    auto end = std::chrono::steady_clock::now();
    long duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
    printf("Elapsed time for copying file  ms : %li ms.\n\n", duration);

    fclose(file);
    
    for(size_t i = 0; i < 10; i++){
        printf("trip_seconds[%d]: %f, trip_miles: %f fare_amount: %f tip_amount: %f tolls: %f extra: %f\n", 
                i, trip_seconds[i], trip_miles[i], fare_amount[i], 
                   tip_amount[i], tolls[i], extra[i]);
    }

    printf("1\n");

    size_t number = 0;
    for(size_t i = 0; i < max; i++){

        if(trip_seconds[i] > THRESHOLD_SECONDS){
            number++;
        }
    }

    printf("number: %llu\n", number);

    printf("size1: %llu\n", size);

    for(size_t i = 0; i < max; i++){
        seconds[i] = trip_seconds[i];
        miles[i] = trip_miles[i];
        fare[i] = fare_amount[i];
        extra[i] = tip_amount[i];
        tips[i] = tolls_amount[i];
        tolls[i] = extra_amount[i];
    }

    free(trip_seconds);
    free(trip_miles);
    free(fare_amount);
    free(tip_amount);
    free(tolls_amount);
    free(extra_amount);

    printf("line: %d\n", __LINE__);

    return 0;

}

int rapids_uvm_direct(char *filename){

    printf("Started processing 0.\n");
    size_t count = 0;
    
    // file = fopen(filename, "r");
    
    float *seconds, *miles, *fare, *extra, *tips, *tolls;
    read_bin_uvm(filename, count, seconds, miles, fare, extra, tips, tolls);

    size_t size = count*sizeof(float)*6; // MAX_TRIPS * LINE_LENGTH * sizeof(char);
    printf("size: %llu\n", size);
    
    
    printf("size0: %llu\n", size);
    printf("size1: %llu\n", size);

    printf("size2: %llu\n", size);
    // fclose(file);
    // return 0;

    hipEvent_t event1, event2;
    hipEventCreate(&event1);
    hipEventCreate(&event2);
    hipError_t ret;
    
    float *total_amount, *h_total, *d_miles, *h_miles, *h_sum, *d_sum;
    h_miles = (float *) malloc(sizeof(float)*count);
    h_sum = (float *) malloc(sizeof(float)*count);
    check_cuda_error(hipMalloc((void **)&d_sum, sizeof(float)*count));
    for(size_t i = 0; i < count; i++) {
        h_sum[i] = 0;
        h_miles[i] = 0;
    }
    check_cuda_error(hipMemcpy(d_sum, h_sum, sizeof(float)*count, hipMemcpyHostToDevice));

    check_cuda_error(hipMalloc((void **)&d_miles, sizeof(float)*count));
    check_cuda_error(hipMemcpy(d_miles, h_miles, sizeof(float)*count, hipMemcpyHostToDevice));
    // check_cuda_error(hipMemset(d_miles, 0, sizeof(float)));
    
    h_total = (float *) malloc(sizeof(float));

    // Step 1: Read CSV into a single buffer
    // if (read_csv(file, buffer, &count) != 0) {
    //     return -1;
    // }

    printf("count: %llu\n", count);

    int *h_array, *d_array, h_ones, *d_ones; 
    h_array = (int *) malloc(sizeof(int)*count);

    for(size_t i = 0; i < count; i++) h_array[i] = 0;

    check_cuda_error(hipMalloc((void **)&d_ones, sizeof(int)));
    check_cuda_error(hipMemset(d_ones, 0, sizeof(int)));

    check_cuda_error(hipMalloc((void **)&d_array, sizeof(int)*count));
    check_cuda_error(hipMemcpy(d_array, h_array, sizeof(int)*count, hipMemcpyHostToDevice));

    
    
    
    check_cuda_error(hipMalloc((void **)&total_amount, sizeof(float)));
    check_cuda_error(hipMemset(total_amount, 0, sizeof(float)));
    

    ret = hipDeviceSynchronize();
    hipEventRecord(event1, (hipStream_t)1);

    // Step 3: Copy data to device
    // check_cuda_error(hipMemcpy(d_buffer, buffer, count * LINE_LENGTH * sizeof(char), hipMemcpyHostToDevice));

    // Step 4: Launch kernel
    size_t n_pages = size/(REQUEST_SIZE*6);
    int threadsPerBlock = 1024;
    int blocksPerGrid = (count + threadsPerBlock - 1) / threadsPerBlock;
    ret = hipDeviceSynchronize();
    printf("ret: %d hipGetLastError(): %d\n", ret, hipGetLastError());
    auto start = std::chrono::steady_clock::now();
    process_trips_uvm_direct_trip_miles2<<</*(n_pages*WARP_SIZE)/threadsPerBlock+1, threadsPerBlock */blocksPerGrid, threadsPerBlock >>>
                    (seconds, miles, fare, extra, tips, tolls, n_pages, count, total_amount, d_miles, d_array, d_sum);
    ret = hipDeviceSynchronize();
    printf("ret: %d hipGetLastError(): %d\n", ret, hipGetLastError());
    auto end = std::chrono::steady_clock::now();
    long duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
    printf("Elapsed time for trip_miles uvm transfer  ms : %li ms.\n\n", duration);

    // start = std::chrono::steady_clock::now();
    // process_trips_rdma_direct_fare_amount<<< (n_pages*32)/threadsPerBlock+1, threadsPerBlock /*blocksPerGrid, threadsPerBlock*/ >>>
    //                 (rdma_buffer_global, n_pages, count, total_amount, d_miles, d_array, d_ones);
    // ret = hipDeviceSynchronize();
    // printf("ret: %d hipGetLastError(): %d\n", ret, hipGetLastError());
    // end = std::chrono::steady_clock::now();
    // duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
    // printf("Elapsed time for fare_amount rdma transfer  ms : %li ms.\n\n", duration);

    // start = std::chrono::steady_clock::now();
    // process_trips_rdma_direct_tip_amount<<< blocksPerGrid, threadsPerBlock >>>(rdma_buffer_global, count, total_amount, d_miles);
    // ret = hipDeviceSynchronize();
    // printf("ret: %d hipGetLastError(): %d\n", ret, hipGetLastError());
    // end = std::chrono::steady_clock::now();
    // duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
    // printf("Elapsed time for tip_amount rdma transfer  ms : %li ms.\n\n", duration);

    // start = std::chrono::steady_clock::now();
    // process_trips_rdma_direct_tolls<<< blocksPerGrid, threadsPerBlock >>>(rdma_buffer_global, count, total_amount, d_miles);
    // ret = hipDeviceSynchronize();
    // printf("ret: %d hipGetLastError(): %d\n", ret, hipGetLastError());
    // end = std::chrono::steady_clock::now();
    // duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
    // printf("Elapsed time for tolls rdma transfer  ms : %li ms.\n\n", duration);

    // start = std::chrono::steady_clock::now();
    // process_trips_rdma_direct_extra<<< blocksPerGrid, threadsPerBlock >>>(rdma_buffer_global, count, total_amount, d_miles);
    // ret = hipDeviceSynchronize();
    // printf("ret: %d hipGetLastError(): %d\n", ret, hipGetLastError());
    // end = std::chrono::steady_clock::now();
    // duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
    // printf("Elapsed time for extra rdma transfer  ms : %li ms.\n\n", duration);

    
    hipDeviceSynchronize();

    hipEventRecord(event2, (hipStream_t) 1);
            
    hipEventSynchronize(event1); //optional
    hipEventSynchronize(event2); //wait for the event to be executed!

    ret = hipDeviceSynchronize();

    print_retires<<<1,1>>>();

    check_cuda_error(hipMemcpy(h_array, d_array, sizeof(int)*count, hipMemcpyDeviceToHost));
    check_cuda_error(hipMemcpy(h_total, total_amount, sizeof(float), hipMemcpyDeviceToHost));
    check_cuda_error(hipMemcpy(h_miles, d_miles, sizeof(float)*count, hipMemcpyDeviceToHost));
    check_cuda_error(hipMemcpy(&h_ones, d_ones, sizeof(int), hipMemcpyDeviceToHost));
    check_cuda_error(hipMemcpy(h_sum, d_sum, sizeof(float)*count, hipMemcpyDeviceToHost));
    
    float dt_ms;
    hipEventElapsedTime(&dt_ms, event1, event2);
    printf("The cudaEvent execution time with rdma on GPU: %f ms\n", dt_ms);

    size_t ones = 0;
    float sum = 0;
    float sum_miles = 0;
    for(size_t i = 0; i < count; i++) {
        if(h_array[i] == 1)
            ones++;
        sum += h_sum[i];
        sum_miles += h_miles[i];

    }

    printf("sum: %f\n", sum);
    printf("atomic ones: %d\n", h_ones);
    printf("ones: %d\n", ones);
    printf("h_total: %f\n", *h_total);
    printf("sum_miles: %f\n", sum_miles);
    printf("Avg. $/mile: %f\n", sum/sum_miles);

    // Step 5: Free memory
    // free(buffer);
    // hipFree(d_buffer);

    return 0;
}

int filter_bin(char *filename){
    size_t max = 211670894llu;
    // max1 = max;
    float *trip_seconds; // [MAX_TRIPS];
    float *trip_miles;
    float *fare_amount;
    float *tip_amount;
    float *tolls;
    float *extra;
    trip_seconds = (float *) malloc(sizeof(float)*max);
    trip_miles = (float *) malloc(sizeof(float)*max);
    fare_amount = (float *) malloc(sizeof(float)*max);
    tip_amount = (float *) malloc(sizeof(float)*max);
    tolls = (float *) malloc(sizeof(float)*max);
    extra = (float *) malloc(sizeof(float)*max);
    
    

    size_t res;
    size_t size = max*sizeof(float)*6;
    // check_cuda_error(hipMallocManaged((void **)&rdma_buffer_global, sizeof(rdma_buf<float>)));
    auto start = std::chrono::steady_clock::now();
    
    FILE *file = fopen(filename, "rb");
    if (file == NULL) {
        perror("Failed to open file for reading");
        return EXIT_FAILURE;
    }

    res = fread(trip_seconds, sizeof(float), max, file);
    if (res != max) {
        perror("Failed to read all data in trip_seconds");
    }

    res = fread(trip_miles, sizeof(float), max, file);
    if (res != max) {
        perror("Failed to read all data in trip_miles");
    }
    
    res = fread(fare_amount, sizeof(float), max, file);
    if (res != max) {
        perror("Failed to read all data in fare_amount");
    }
    
    res = fread(tip_amount, sizeof(float), max, file);
    if (res != max) {
        perror("Failed to read all data in tip_amount");
    }
    
    res = fread(tolls, sizeof(float), max, file);
    if (res != max) {
        perror("Failed to read all data in tolls");
    }
    
    res = fread(extra, sizeof(float), max, file);
    if (res != max) {
        perror("Failed to read all data in extra");
    }

    auto end = std::chrono::steady_clock::now();
    long duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
    printf("Elapsed time for copying file  ms : %li ms.\n\n", duration);

    fclose(file);
    
    for(size_t i = 0; i < 10; i++){
        printf("trip_seconds[%d]: %f, trip_miles: %f fare_amount: %f tip_amount: %f tolls: %f extra: %f\n", 
                i, trip_seconds[i], trip_miles[i], fare_amount[i], 
                   tip_amount[i], tolls[i], extra[i]);
    }

    printf("1\n");

    size_t number = 0;
    float sum_fare = 0;
    for(size_t i = 0; i < max; i++){

        if(trip_seconds[i] > THRESHOLD_SECONDS){
            number++;
            sum_fare += fare_amount[i];
        }

        // sum_fare += fare_amount[i];


    }

    printf("sum_fare: %f\n", sum_fare);

    free(trip_seconds);
    free(trip_miles);
    free(fare_amount);
    free(tip_amount);
    free(tolls);
    free(extra);

    printf("number: %llu\n", number);

    printf("size1: %llu\n", size);

    return 0;

}

__global__ __launch_bounds__(1024,2)
void process_trips_rdma_direct(rdma_buf<float> *rdma_file, int count, float *total_amount, float *total_miles) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < count) {
        
        // Declare an array to hold the line
        // char line[LINE_LENGTH];  // Create a character array for one line

        // int vendorID;
        float trip_seconds = (*rdma_file)[idx];
        
        // parse_trip(line, trip_seconds, trip_miles, fare_amount, tip_amount, tolls, extra);
        // trip_miles < 1050

        if(trip_seconds > 1000) {

            // printf("trip_seconds: %f\n", trip_seconds);

            float trip_miles  = (*rdma_file)[5*idx  + count + 0];
            float fare_amount = (*rdma_file)[5*idx + count + 1];
            float tip_amount  = (*rdma_file)[5*idx + count + 2];
            float tolls       = (*rdma_file)[5*idx + count + 3];
            float extra       = (*rdma_file)[5*idx + count + 4];

            float local_total = fare_amount; // - extra - tolls + tip_amount;
            
                AtomicAdd(total_amount, local_total);
                AtomicAdd(total_miles, trip_miles);
        }

        // Example operation: Store or print the parsed values (printing from device is not recommended)
        // printf("Vendor ID: %d, Pickup Latitude: %f, Pickup Longitude: %f\n", vendorID, pickup_latitude, pickup_longitude);
    }
}

__global__ // __launch_bounds__(1024,2)
void process_trips_rdma_direct_trip_miles(rdma_buf<float> *rdma_file, size_t n, size_t count, float *total_amount, float *total_miles, int *array, float *sum) {


    // Page size in elements (64KB / 4 bytes per unsigned int)
    const size_t pageSize = REQUEST_SIZE / sizeof(float);
    // Elements per warp
    const size_t elementsPerWarp = pageSize / WARP_SIZE;

    // Global thread ID
    size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    // if(tid == 0) printf("warpSize: %d\n", warpSize);
    // Warp ID within the block
    size_t warpId = tid / WARP_SIZE;

    // Thread lane within the warp
    size_t lane = threadIdx.x % WARP_SIZE; // warpSize;

    // Determine which page this warp will process
    size_t pageStart = warpId * pageSize;

    // Ensure we don't process out-of-bounds pages
    if (pageStart < n * pageSize) {
        
        // Process elements within the page
        // for (size_t i = 0; i < elementsPerWarp; ++i) {
        //     size_t elementIdx = pageStart + lane + i * warpSize;
            size_t end = (warpId + 1)*pageSize > count ? count : (warpId + 1)*pageSize;
            for(size_t j = warpId*pageSize + lane; j < end; j += WARP_SIZE) {
                float trip_seconds = (*rdma_file)[j];
                if(trip_seconds > THRESHOLD_SECONDS) {
                    // 
                    // if(trip_miles < 3460.0){
                        // size_t index = 5*j + count;
                        // float trip_miles  = (*rdma_file)[index ];
                        // float fare_amount = (*rdma_file)[index + 1];     
                        // float extra       = (*rdma_file)[index + 4];
                        // float tip_amount  = (*rdma_file)[index + 2];
                        // float tolls       = (*rdma_file)[index + 3];

                        float trip_miles  = (*rdma_file)[j + count];
                        float fare_amount = (*rdma_file)[j + count*2];
                        
                        float extra       = (*rdma_file)[j + count*5];
                        float tip_amount  = (*rdma_file)[j + count*3];
                        float tolls       = (*rdma_file)[j + count*4];
                        
                        

                        float local_total = fare_amount - extra + tip_amount - tolls;
                        sum[j] += local_total;
                        total_miles[j] += trip_miles;
                        
                        atomicAdd(total_amount, local_total);
                        atomicAdd(total_miles, trip_miles);
                        // array[j] = 1;
                        // atomicAdd(ones, 1);
                    // }
                }
                // else{
                //     printf("trip_seconds: %f\n", trip_seconds);
                // }
            }
        // }
    }


    // size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    // if (idx < count) {
    //     float trip_seconds = (*rdma_file)[idx];
    //     if(trip_seconds > 1000) {
    //         float trip_miles  = (*rdma_file)[5*idx  + count + 0];
    //         // float fare_amount = (*rdma_file)[5*idx + count + 1];
    //         // float tip_amount  = (*rdma_file)[5*idx + count + 2];
    //         // float tolls       = (*rdma_file)[5*idx + count + 3];
    //         // float extra       = (*rdma_file)[5*idx + count + 4];

    //         // float local_total = fare_amount - extra - tolls + tip_amount;
            
    //         //     AtomicAdd(total_amount, local_total);
    //             AtomicAdd(total_miles, trip_miles);
    //     }
    // }
}


__global__ __launch_bounds__(1024,2)
void process_trips_rdma_direct_fare_amount(rdma_buf<float> *rdma_file, size_t n, size_t count, float *total_amount, float *total_miles, int *array, int *ones) {

    // Page size in elements (64KB / 4 bytes per unsigned int)
    const size_t pageSize = REQUEST_SIZE / sizeof(float);
    // Elements per warp
    const size_t elementsPerWarp = pageSize / warpSize;

    // Global thread ID
    size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    // if(tid == 0) printf("warpSize: %d\n", warpSize);
    // Warp ID within the block
    size_t warpId = tid / warpSize;

    // Thread lane within the warp
    size_t lane = threadIdx.x % warpSize; // warpSize;

    // Determine which page this warp will process
    size_t pageStart = warpId * pageSize;

    // Ensure we don't process out-of-bounds pages
    if (pageStart < n * pageSize) {
        
        // Process elements within the page
        // for (size_t i = 0; i < elementsPerWarp; ++i) {
        //     size_t elementIdx = pageStart + lane + i * warpSize;
            size_t end = (warpId + 1)*pageSize > count ? count : (warpId + 1)*pageSize;
            for(size_t j = warpId*pageSize + lane; j < end; j += warpSize) {
                
                if(array[j] == 1) {
                    float fare_amount = (*rdma_file)[j + count*2];
                    // atomicAdd(total_amount, fare_amount);
                    atomicAdd(ones, 1);
                }
                
            }
        // }
    }

    // size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    // if (idx < count) {
    //     // float trip_seconds = (*rdma_file)[idx];
    //     if(array[idx]) {
    //         // float fare_amount  = (*rdma_file)[5*idx  + count + 1];
    //         float fare_amount = (*rdma_file)[idx + count*2];
            
    //             
    //             // AtomicAdd(total_miles, trip_miles);
    //     }
    // }
}

__global__ __launch_bounds__(1024,2)
void process_trips_rdma_direct_tip_amount(rdma_buf<float> *rdma_file, size_t count, float *total_amount, float *total_miles) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < count) {
        float trip_seconds = (*rdma_file)[idx];
        if(trip_seconds > THRESHOLD_SECONDS) {
            // float trip_miles  = (*rdma_file)[5*idx  + count + 0];
            // float fare_amount = (*rdma_file)[5*idx + count + 1];
            float tip_amount  = (*rdma_file)[idx + count*3];
            // float tolls       = (*rdma_file)[5*idx + count + 3];
            // float extra       = (*rdma_file)[5*idx + count + 4];

            // float local_total = fare_amount - extra - tolls + tip_amount;
            
                atomicAdd(total_amount, tip_amount);
                // AtomicAdd(total_miles, trip_miles);
        }
    }
}

__global__ __launch_bounds__(1024,2)
void process_trips_rdma_direct_tolls(rdma_buf<float> *rdma_file, size_t count, float *total_amount, float *total_miles) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < count) {
        float trip_seconds = (*rdma_file)[idx];
        if(trip_seconds > THRESHOLD_SECONDS) {
            // float trip_miles  = (*rdma_file)[5*idx  + count + 0];
            // float fare_amount = (*rdma_file)[5*idx + count + 1];
            // float tip_amount  = (*rdma_file)[5*idx + count + 2];
            float tolls       = (*rdma_file)[idx + count*4] * (-1);
            // float extra       = (*rdma_file)[5*idx + count + 4];

            // float local_total = fare_amount - extra - tolls + tip_amount;
            
                atomicAdd(total_amount, tolls);
                // AtomicAdd(total_miles, trip_miles);
        }
    }
}

__global__ __launch_bounds__(1024,2)
void process_trips_rdma_direct_extra(rdma_buf<float> *rdma_file, size_t count, float *total_amount, float *total_miles) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < count) {
        float trip_seconds = (*rdma_file)[idx];
        if(trip_seconds > THRESHOLD_SECONDS) {
            // float trip_miles  = (*rdma_file)[5*idx  + count + 0];
            // float fare_amount = (*rdma_file)[5*idx + count + 1];
            // float tip_amount  = (*rdma_file)[5*idx + count + 2];
            // float tolls       = (*rdma_file)[5*idx + count + 3] * (-1);
            float extra       = (*rdma_file)[idx + count*5] * (-1);

            // float local_total = fare_amount - extra - tolls + tip_amount;
            
                atomicAdd(total_amount, extra);
                // AtomicAdd(total_miles, trip_miles);
        }
    }
}

int write_bin(char *filename, char *bin_file){
    size_t max = 211670894*2;
    // char line1[LINE_LENGTH];
    // FILE *file1 = fopen(filename, "r");
    // if (!file1) {
    //     perror("Failed to open file rapids_RDMA_direct");
    //     return -1;
    // }

    // while (fgets(line1, LINE_LENGTH, file1)){
    //     max++;
    // }
    // printf("max: %llu\n", max);

    // fclose(file1);


    float *trip_seconds; // [MAX_TRIPS];
    float *trip_miles;
    float *fare_amount;
    float *tip_amount;
    float *tolls;
    float *extra;
    trip_seconds = (float *) malloc(sizeof(float)*max);
    trip_miles = (float *) malloc(sizeof(float)*max);
    fare_amount = (float *) malloc(sizeof(float)*max);
    tip_amount = (float *) malloc(sizeof(float)*max);
    tolls = (float *) malloc(sizeof(float)*max);
    extra = (float *) malloc(sizeof(float)*max);
    if(trip_seconds == NULL || trip_miles == NULL || fare_amount == NULL ||
       tip_amount == NULL || tolls == NULL || extra == NULL ){
        printf("error on allocation\n");
        return -1;
    }
    char *line; //[LINE_LENGTH];
    size_t i_seconds = 0, i_miles = 0, i_fare = 0, i_tip = 0, i_tolls = 0, i_extra = 0;
    printf("Started processing 1.\n");
    
    // char *buffer;
    FILE *file = fopen(filename, "r");
    if (!file) {
        perror("Failed to open file rapids_RDMA_direct");
        return -1;
    }
    printf("Started processing.\n");
    size_t counter_line = 0;
    size_t count = 0, len = 0;
    while (getline(&line, &len, file) != -1/* && count < MAX_ROWS*/) {
        if (counter_line == 0) {
            counter_line++;
            continue;  // Skip the header line
        }

        char small_buffer[50]; // Temporary buffer to store individual field data
        size_t field = 0; // Field index
        size_t index = 0; // Index for small_buffer
        
        // Iterate through the line character by character
        for (size_t i = 0; i < len; i++) {
            if (line[i] == ',' || line[i] == '\n') { // Check for comma or newline
                small_buffer[index] = '\0'; // Null-terminate the string

                // Check for empty fields (NA) or invalid values
                if (index == 0 || strcmp(small_buffer, "") == 0) {
                    strcpy(small_buffer, "0"); // Replace with "0" if empty
                }

                // Store the value in the appropriate buffer
                if (field == 4) {
                    
                    trip_seconds[i_seconds] = atof(small_buffer); // Convert to float
                    if(i_seconds < 10) printf("i_seconds: %d small_buffer: %s trip_seconds[i_seconds]; %f\n", i_seconds, small_buffer, trip_seconds[i_seconds]);
                    // printf(" small_buffer: %s trip_seconds[%llu]: %f ", small_buffer, i_seconds, trip_seconds[i_seconds]);
                    i_seconds++;
                } else if (field == 5) {
                    trip_miles[i_miles] = atof(small_buffer);
                    i_miles++;
                } else if (field == 10) {
                    fare_amount[i_fare] = atof(small_buffer);
                    i_fare++;
                } else if (field == 11) {
                    tip_amount[i_tip] = atof(small_buffer);
                    i_tip++;
                } else if (field == 12) {
                    tolls[i_tolls] = atof(small_buffer);
                    i_tolls++;
                } else if (field == 13) {
                    extra[i_extra] = atof(small_buffer);
                    i_extra++; // Increment count after the last field
                }
                
                field++; // Move to the next field
                index = 0; // Reset index for the next field
            } else {
                small_buffer[index++] = line[i]; // Collect character into buffer
            }
        }
        
        counter_line++;
    
        // if(count < 10){
        //     printf("trip_seconds[%d]: %f ", count, trip_seconds[count]);
        //     printf("trip_miles[%d]: %f ", count, trip_miles[count]);
        //     printf("fare_amount[%d]: %f ", count, fare_amount[count]);
        //     printf("tip_amount[%d]: %f ", count, tip_amount[count]);
        //     printf("tolls[%d]: %f ", count, tolls[count]);
        //     printf("extra[%d]: %f\n", count, extra[count]);
        // }

    }

    free(line); // Free the line buffer
    fclose(file);

    printf("i_seconds: %d, i_miles: %d i_fare: %d i_tip: %d i_tolls: %d i_extra: %d max; %d\n", 
                i_seconds, i_miles, i_fare, i_tip, i_tolls, i_extra, max);

    for(size_t i = 0; i < 10; i++){
        printf("trip_seconds[%d]: %f, trip_miles: %f fare_amount: %f tip_amount: %f tolls: %f extra: %f\n", 
                i, trip_seconds[i], trip_miles[i], fare_amount[i], tip_amount[i], tolls[i], extra[i]);
    }

    // Assume we have two columns of data: column1 and column2
    FILE *f = fopen(bin_file, "wb");

    printf("count: %llu\n", count);

    // Write column1 data (assume it's an array of floats)
    size_t res = fwrite(trip_seconds, sizeof(float), i_seconds, f);
    if (res != i_seconds) {
        perror("Failed to write all data in trip_seconds");
    }
    printf("Trip Seconds written\n");
    res = fwrite(trip_miles, sizeof(float), i_miles, f);
    if (res != i_miles) {
        perror("Failed to write all data in trip_miles");
    }
    printf("Trip Miles written\n");
    res = fwrite(fare_amount, sizeof(float), i_fare, f);
    if (res != i_fare) {
        perror("Failed to write all data in fare_amount");
    }
    printf("Fare written\n");
    res = fwrite(tip_amount, sizeof(float), i_tip, f);
    if (res != i_tip) {
        perror("Failed to write all data in tip_amount");
    }
    printf("Tip amounts written\n");
    res = fwrite(tolls, sizeof(float), i_tolls, f);
    if (res != i_tolls) {
        perror("Failed to write all data in tolls");
    }
    printf("Tolls written\n");
    res = fwrite(extra, sizeof(float), i_extra, f);
    if (res != i_extra) {
        perror("Failed to write all data in extra");
    }
    printf("Extra written\n");

    fclose(f);

    printf("File is closed: %s\n", bin_file);

    free(trip_seconds);
    free(trip_miles);
    free(fare_amount);
    free(tip_amount);
    free(tolls);
    free(extra);

    return 0;
}

rdma_buf<float> *rdma_buffer_global = NULL;

int read_bin(char *filename, size_t &max1){
    size_t max = 211670894llu;
    max1 = max;
    float *trip_seconds; // [MAX_TRIPS];
    float *trip_miles;
    float *fare_amount;
    float *tip_amount;
    float *tolls;
    float *extra;
    trip_seconds = (float *) malloc(sizeof(float)*max);
    trip_miles = (float *) malloc(sizeof(float)*max);
    fare_amount = (float *) malloc(sizeof(float)*max);
    tip_amount = (float *) malloc(sizeof(float)*max);
    tolls = (float *) malloc(sizeof(float)*max);
    extra = (float *) malloc(sizeof(float)*max);
    

    size_t res;
    size_t size = max*sizeof(float)*6;
    check_cuda_error(hipMallocManaged((void **)&rdma_buffer_global, sizeof(rdma_buf<float>)));

    rdma_buffer_global->start(size, GPU, NULL);

    auto start = std::chrono::steady_clock::now();
    
    FILE *file = fopen(filename, "rb");
    if (file == NULL) {
        perror("Failed to open file for reading");
        return EXIT_FAILURE;
    }

    res = fread(trip_seconds, sizeof(float), max, file);
    if (res != max) {
        perror("Failed to read all data in trip_seconds");
    }

    res = fread(trip_miles, sizeof(float), max, file);
    if (res != max) {
        perror("Failed to read all data in trip_miles");
    }
    
    res = fread(fare_amount, sizeof(float), max, file);
    if (res != max) {
        perror("Failed to read all data in fare_amount");
    }
    
    res = fread(tip_amount, sizeof(float), max, file);
    if (res != max) {
        perror("Failed to read all data in tip_amount");
    }
    
    res = fread(tolls, sizeof(float), max, file);
    if (res != max) {
        perror("Failed to read all data in tolls");
    }
    
    res = fread(extra, sizeof(float), max, file);
    if (res != max) {
        perror("Failed to read all data in extra");
    }

    auto end = std::chrono::steady_clock::now();
    long duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
    printf("Elapsed time for copying file  ms : %li ms.\n\n", duration);

    fclose(file);
    
    for(size_t i = 0; i < 10; i++){
        printf("trip_seconds[%d]: %f, trip_miles: %f fare_amount: %f tip_amount: %f tolls: %f extra: %f\n", 
                i, trip_seconds[i], trip_miles[i], fare_amount[i], 
                   tip_amount[i], tolls[i], extra[i]);
    }

    printf("1\n");

    size_t number = 0;
    for(size_t i = 0; i < max; i++){

        if(trip_seconds[i] > THRESHOLD_SECONDS){
            number++;
        }
    }

    printf("number: %llu\n", number);

    printf("size1: %llu\n", size);

    for(size_t i = 0; i < max; i++){
        rdma_buffer_global->local_buffer[i] = trip_seconds[i];

        // rdma_buffer_global->local_buffer[5*i + max + 0] = trip_miles[i];
        // rdma_buffer_global->local_buffer[5*i + max + 1] = fare_amount[i];
        // rdma_buffer_global->local_buffer[5*i + max + 2] = tip_amount[i];
        // rdma_buffer_global->local_buffer[5*i + max + 3] = tolls[i];
        // rdma_buffer_global->local_buffer[5*i + max + 4] = extra[i];

        rdma_buffer_global->local_buffer[i + max] = trip_miles[i];
        rdma_buffer_global->local_buffer[i + max*2] = fare_amount[i];
        rdma_buffer_global->local_buffer[i + max*3] = tip_amount[i];
        rdma_buffer_global->local_buffer[i + max*4] = tolls[i];
        rdma_buffer_global->local_buffer[i + max*5] = extra[i];
    }

    free(trip_seconds);
    free(trip_miles);
    free(fare_amount);
    free(tip_amount);
    free(tolls);
    free(extra);

    printf("line: %d\n", __LINE__);

    return 0;

}

int rapids_RDMA_direct(char *filename){

    printf("Started processing 0.\n");
    size_t count = 0;
    
    // file = fopen(filename, "r");
    
    read_bin(filename, count);

    size_t size = count*sizeof(float)*6; // MAX_TRIPS * LINE_LENGTH * sizeof(char);
    printf("size: %llu\n", size);
    
    
    printf("size0: %llu\n", size);
    printf("size1: %llu\n", size);

    printf("size2: %llu\n", size);
    // fclose(file);
    // return 0;

    hipEvent_t event1, event2;
    hipEventCreate(&event1);
    hipEventCreate(&event2);
    hipError_t ret;
    
    float *total_amount, *h_total, *d_miles, *h_miles, *h_sum, *d_sum;
    h_miles = (float *) malloc(sizeof(float)*count);
    h_sum = (float *) malloc(sizeof(float)*count);
    check_cuda_error(hipMalloc((void **)&d_sum, sizeof(float)*count));
    for(size_t i = 0; i < count; i++) {
        h_sum[i] = 0;
        h_miles[i] = 0;
    }
    check_cuda_error(hipMemcpy(d_sum, h_sum, sizeof(float)*count, hipMemcpyHostToDevice));

    check_cuda_error(hipMalloc((void **)&d_miles, sizeof(float)*count));
    check_cuda_error(hipMemcpy(d_miles, h_miles, sizeof(float)*count, hipMemcpyHostToDevice));
    // check_cuda_error(hipMemset(d_miles, 0, sizeof(float)));
    
    h_total = (float *) malloc(sizeof(float));

    // Step 1: Read CSV into a single buffer
    // if (read_csv(file, buffer, &count) != 0) {
    //     return -1;
    // }

    printf("count: %llu\n", count);

    int *h_array, *d_array, h_ones, *d_ones; 
    h_array = (int *) malloc(sizeof(int)*count);

    for(size_t i = 0; i < count; i++) h_array[i] = 0;

    check_cuda_error(hipMalloc((void **)&d_ones, sizeof(int)));
    check_cuda_error(hipMemset(d_ones, 0, sizeof(int)));

    check_cuda_error(hipMalloc((void **)&d_array, sizeof(int)*count));
    check_cuda_error(hipMemcpy(d_array, h_array, sizeof(int)*count, hipMemcpyHostToDevice));

    
    
    
    check_cuda_error(hipMalloc((void **)&total_amount, sizeof(float)));
    check_cuda_error(hipMemset(total_amount, 0, sizeof(float)));
    

    ret = hipDeviceSynchronize();
    hipEventRecord(event1, (hipStream_t)1);

    // Step 3: Copy data to device
    // check_cuda_error(hipMemcpy(d_buffer, buffer, count * LINE_LENGTH * sizeof(char), hipMemcpyHostToDevice));

    // Step 4: Launch kernel
    size_t n_pages = size/(REQUEST_SIZE*6);
    int threadsPerBlock = 1024;
    int blocksPerGrid = (count + threadsPerBlock - 1) / threadsPerBlock;
    ret = hipDeviceSynchronize();
    printf("ret: %d hipGetLastError(): %d\n", ret, hipGetLastError());
    auto start = std::chrono::steady_clock::now();
    process_trips_rdma_direct_trip_miles<<< (n_pages*WARP_SIZE)/threadsPerBlock+1, threadsPerBlock /*blocksPerGrid, threadsPerBlock*/ >>>
                    (rdma_buffer_global, n_pages, count, total_amount, d_miles, d_array, d_sum);
    ret = hipDeviceSynchronize();
    printf("ret: %d hipGetLastError(): %d\n", ret, hipGetLastError());
    auto end = std::chrono::steady_clock::now();
    long duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
    printf("Elapsed time for trip_miles rdma transfer  ms : %li ms.\n\n", duration);

    // start = std::chrono::steady_clock::now();
    // process_trips_rdma_direct_fare_amount<<< (n_pages*32)/threadsPerBlock+1, threadsPerBlock /*blocksPerGrid, threadsPerBlock*/ >>>
    //                 (rdma_buffer_global, n_pages, count, total_amount, d_miles, d_array, d_ones);
    // ret = hipDeviceSynchronize();
    // printf("ret: %d hipGetLastError(): %d\n", ret, hipGetLastError());
    // end = std::chrono::steady_clock::now();
    // duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
    // printf("Elapsed time for fare_amount rdma transfer  ms : %li ms.\n\n", duration);

    // start = std::chrono::steady_clock::now();
    // process_trips_rdma_direct_tip_amount<<< blocksPerGrid, threadsPerBlock >>>(rdma_buffer_global, count, total_amount, d_miles);
    // ret = hipDeviceSynchronize();
    // printf("ret: %d hipGetLastError(): %d\n", ret, hipGetLastError());
    // end = std::chrono::steady_clock::now();
    // duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
    // printf("Elapsed time for tip_amount rdma transfer  ms : %li ms.\n\n", duration);

    // start = std::chrono::steady_clock::now();
    // process_trips_rdma_direct_tolls<<< blocksPerGrid, threadsPerBlock >>>(rdma_buffer_global, count, total_amount, d_miles);
    // ret = hipDeviceSynchronize();
    // printf("ret: %d hipGetLastError(): %d\n", ret, hipGetLastError());
    // end = std::chrono::steady_clock::now();
    // duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
    // printf("Elapsed time for tolls rdma transfer  ms : %li ms.\n\n", duration);

    // start = std::chrono::steady_clock::now();
    // process_trips_rdma_direct_extra<<< blocksPerGrid, threadsPerBlock >>>(rdma_buffer_global, count, total_amount, d_miles);
    // ret = hipDeviceSynchronize();
    // printf("ret: %d hipGetLastError(): %d\n", ret, hipGetLastError());
    // end = std::chrono::steady_clock::now();
    // duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
    // printf("Elapsed time for extra rdma transfer  ms : %li ms.\n\n", duration);

    
    hipDeviceSynchronize();

    hipEventRecord(event2, (hipStream_t) 1);
            
    hipEventSynchronize(event1); //optional
    hipEventSynchronize(event2); //wait for the event to be executed!

    ret = hipDeviceSynchronize();

    print_retires<<<1,1>>>();

    check_cuda_error(hipMemcpy(h_array, d_array, sizeof(int)*count, hipMemcpyDeviceToHost));
    check_cuda_error(hipMemcpy(h_total, total_amount, sizeof(float), hipMemcpyDeviceToHost));
    check_cuda_error(hipMemcpy(h_miles, d_miles, sizeof(float)*count, hipMemcpyDeviceToHost));
    check_cuda_error(hipMemcpy(&h_ones, d_ones, sizeof(int), hipMemcpyDeviceToHost));
    check_cuda_error(hipMemcpy(h_sum, d_sum, sizeof(float)*count, hipMemcpyDeviceToHost));
    
    float dt_ms;
    hipEventElapsedTime(&dt_ms, event1, event2);
    printf("The cudaEvent execution time with rdma on GPU: %f ms\n", dt_ms);

    size_t ones = 0;
    float sum = 0;
    float sum_miles = 0;
    for(size_t i = 0; i < count; i++) {
        if(h_array[i] == 1)
            ones++;
        sum += h_sum[i];
        sum_miles += h_miles[i];

    }

    printf("sum: %f\n", sum);
    printf("atomic ones: %d\n", h_ones);
    printf("ones: %d\n", ones);
    printf("h_total: %f\n", *h_total);
    printf("sum_miles: %f\n", sum_miles);
    printf("Avg. $/mile: %f\n", sum/sum_miles);

    // Step 5: Free memory
    // free(buffer);
    // hipFree(d_buffer);

    return 0;
}


// Main program
int main(int argc, char **argv)
{   
    init_gpu(0);
    hipSetDevice(0);
    printf("hello from rapid\n");
    char *file = argv[7];
    printf("hello from rapid file : %s\n", file);
    // rapids_CUDA(file);
    
    
    // char *bin_file = "/mydata/chicago_2b_trial.bin";
    // write_bin(file, bin_file);

    // printf("Binary file is written\n");

    bool rdma_flag = false;
    hipError_t ret1;
    struct context_2nic *s_ctx = (struct context_2nic *)malloc(sizeof(struct context_2nic));
    if(rdma_flag){
        s_ctx->gpu_cq = NULL;
        s_ctx->wqbuf = NULL;
        s_ctx->cqbuf = NULL;
        s_ctx->gpu_qp = NULL;


        int num_msg = (unsigned long) atoi(argv[4]);
        int mesg_size = (unsigned long) atoi(argv[5]);
        int num_bufs = (unsigned long) atoi(argv[6]);

        
        struct post_content post_cont, *d_post, host_post;
        struct poll_content poll_cont, *d_poll, host_poll;
        // struct post_content2 /*post_cont2,*/ *d_post2;
        struct server_content_2nic post_cont2, *d_post2;
        struct host_keys keys;
        struct gpu_memory_info gpu_mem;

        int num_iteration = num_msg;
        s_ctx->n_bufs = num_bufs;

        s_ctx->gpu_buf_size = 26*1024*1024*1024llu; // N*sizeof(int)*3llu;
        s_ctx->gpu_buffer = NULL;

        // // remote connection:
        // int ret = connect(argv[2], s_ctx);

        // local connect
        char *mlx_name = "mlx5_0";
        // int ret = local_connect(mlx_name, s_ctx);
        int ret = local_connect_2nic(mlx_name, s_ctx, 0, GPU);

        mlx_name = "mlx5_2";
        // int ret = local_connect(mlx_name, s_ctx);
        ret = local_connect_2nic(mlx_name, s_ctx, 1, GPU);

        ret = prepare_post_poll_content_2nic(s_ctx, &post_cont, &poll_cont, &post_cont2, \
                                        &host_post, &host_poll, &keys, &gpu_mem);
        if(ret == -1) {
            printf("Post and poll contect creation failed\n");    
            exit(-1);
        }

        printf("alloc synDev ret: %d\n", hipDeviceSynchronize());
        hipSetDevice(GPU);
        alloc_global_cont(&post_cont, &poll_cont, &post_cont2, gpu_mem);

        // if(hipSuccess != ){    
        printf("alloc synDev ret1: %d\n", hipDeviceSynchronize());
            // return -1;
        // }

        ret1 = hipDeviceSynchronize();
        printf("ret: %d\n", ret1);
        if(hipSuccess != ret1){    
            return -1;
        }

        size_t restricted_gpu_mem = 16*1024*1024*1024llu;
        // restricted_gpu_mem = restricted_gpu_mem / 3;
        const size_t page_size = REQUEST_SIZE;
        // const size_t numPages = ceil((double)restricted_gpu_mem/page_size);

        printf("function: %s line: %d\n", __FILE__, __LINE__);
        alloc_global_host_content(host_post, host_poll, keys, gpu_mem);
        printf("function: %s line: %d\n", __FILE__, __LINE__);

        ret1 = hipDeviceSynchronize();
        printf("ret: %d\n", ret1);
        if(hipSuccess != ret1){    
            return -1;
        }

        ret1 = hipDeviceSynchronize();
        printf("ret: %d\n", ret1);
        if(hipSuccess != ret1){    
            return -1;
        }
        
        printf("restricted_gpu_mem: %zu\n", restricted_gpu_mem);
        hipSetDevice(GPU);
        start_page_queue<<<1, 1>>>(/*s_ctx->gpu_buf_size*/restricted_gpu_mem, page_size);
        ret1 = hipDeviceSynchronize();
        printf("ret: %d\n", ret1);
        if(hipSuccess != ret1){    
            return -1;
        }
    }

    
    

    if(rdma_flag){
        // the follwoing for directly reading file in rdma
        // rapids_RDMA(file);
        // following for reading buffers separately
        rapids_RDMA_direct(file);
        // transfer_benchmark();
        hipFree(s_ctx->gpu_buffer);
    }
    else{
        rapids_uvm_direct(file);
    }

    filter_bin(file);

    // rapids_CUDA(file);

    // rapids_CUDA(file);

    // rapids_CUDA(file);
    
    // printf("oversubs ratio: %d\n", oversubs_ratio_macro-1);
    
	return 0;
}