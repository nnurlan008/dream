#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <string>
#include <cstring>
#include <math.h>
#include <chrono>

using namespace std;
// using namespace std;


// extern "C"{
//   #include "rdma_utils.h"
// }

// #include "../../src/rdma_utils.cuh"
#include <time.h>
// #include "../../include/runtime_prefetching.h"
// #include "../../include/runtime_eviction.h"
// #include "../../include/runtime_micro.h"
#include "../../include/runtime_prefetching_2nic.h"


//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.05

//define a small float value
#define SMALL_FLOAT_VAL 0.00000001f

/* Problem size. */
#define NX 4096*16llu
#define NY 4096*16llu

#define BLOCK_NUM 1024ULL
#define MYINFINITY 2147483647llu

#define BLOCK_SIZE 1024
#define WARP_SHIFT 5
#define WARP_SIZE 32

#define GPU 0

typedef float DATA_TYPE;

/* Thread block dimensions */
#define DIM_THREAD_BLOCK_X 1024
#define DIM_THREAD_BLOCK_Y 1

#ifndef M_PI
#define M_PI 3.14159
#endif

__device__ rdma_buf<unsigned int> D_adjacencyList;

__global__ void test(rdma_buf<unsigned int> *a/*, rdma_buf<int> *b, rdma_buf<int> *c*/);


// Kernel
__global__ void add_vectors_uvm(int *a, int *b, int *c, int size)
{
	int id = blockDim.x * blockIdx.x + threadIdx.x;
	// if(id < size) {
		c[id] = a[id] + b[id];
		// printf("c[%d]: %d\n", id, c[id]);
	// }
}

#define htonl(x)  ((((uint32_t)(x) & 0xff000000) >> 24) |\
                   (((uint32_t)(x) & 0x00ff0000) >>  8) |\
                   (((uint32_t)(x) & 0x0000ff00) <<  8) |\
                   (((uint32_t)(x) & 0x000000ff) << 24))

#define WARP_SIZE 32

void delay(int number_of_seconds)
{
    // Converting time into milli_seconds
    int milli_seconds = 1000000 * number_of_seconds;
 
    // Storing start time
    clock_t start_time = clock();
 
    // looping till required time is not achieved
    while (clock() < start_time + milli_seconds)
        ;
}

enum { NS_PER_SECOND = 1000000000 };

void sub_timespec(struct timespec t1, struct timespec t2, struct timespec *td)
{
    td->tv_nsec = t2.tv_nsec - t1.tv_nsec;
    td->tv_sec  = t2.tv_sec - t1.tv_sec;
    if (td->tv_sec > 0 && td->tv_nsec < 0)
    {
        td->tv_nsec += NS_PER_SECOND;
        td->tv_sec--;
    }
    else if (td->tv_sec < 0 && td->tv_nsec > 0)
    {
        td->tv_nsec -= NS_PER_SECOND;
        td->tv_sec++;
    }
}

void usage(const char *argv0)
{
  fprintf(stderr, "usage: %s <mode> <server-address> <server-port>\n  mode = \"read\", \"write\"\n", argv0);
  exit(1);
}


#define check_cuda_error(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }

}

__global__ void transfer(size_t size, rdma_buf<DATA_TYPE> *d_adjacencyList)
{
    size_t id = blockDim.x * blockIdx.x + threadIdx.x;
    size_t stride = blockDim.x * gridDim.x;
    
        for (size_t i = id; i < size ; i += stride)
        {
            DATA_TYPE y = (*d_adjacencyList)[i];
        }
}

__global__ void check(size_t size, rdma_buf<DATA_TYPE> *d_adjacencyList, DATA_TYPE *a)
{
    size_t id = blockDim.x * blockIdx.x + threadIdx.x;
    size_t stride = blockDim.x * gridDim.x;
    
        for (size_t i = id; i < size ; i += stride)
        {
            DATA_TYPE y = (*d_adjacencyList)[i];
            if(a[i] != y){
                printf("y: %f %f ", y, a[i]);
            }
        }
}

__global__ void assign_array(rdma_buf<unsigned int> *adjacencyList){
    D_adjacencyList = *adjacencyList;
    printf("D_adjacencyList.d_TLB[0].state: %d\n", D_adjacencyList.d_TLB[0].state);
    printf("D_adjacencyList.d_TLB[0].device_address: %p\n", D_adjacencyList.d_TLB[0].device_address);
}

int alloc_global_cont(struct post_content *post_cont, struct poll_content *poll_cont, struct server_content_2nic *post_cont2, 
                      struct gpu_memory_info gpu_mem){
    struct post_content *d_post;
    struct poll_content *d_poll;
    struct server_content_2nic *d_post2;

    hipError_t ret0 = hipMalloc((void **)&d_post, sizeof(struct post_content));
    if(ret0 != hipSuccess){
        printf("Error on allocation post content!\n");
        return -1;
    }
    ret0 = hipMalloc((void **)&d_poll, sizeof(struct poll_content));
    if(ret0 != hipSuccess){
        printf("Error on allocation poll content!\n");
        return -1;
    }
    printf("sizeof(struct post_content): %d, sizeof(struct poll_content): %d\n", sizeof(struct post_content), sizeof(struct poll_content));
    ret0 = hipMemcpy(d_post, post_cont, sizeof(struct post_content), hipMemcpyHostToDevice);
    if(ret0 != hipSuccess){
        printf("Error on post copy!\n");
        return -1;
    }
    ret0 = hipMemcpy(d_poll, poll_cont, sizeof(struct poll_content), hipMemcpyHostToDevice);
    if(ret0 != hipSuccess){
        printf("Error on poll copy!\n");
        return -1;
    }

    ret0 = hipMalloc((void **)&d_post2, sizeof(struct server_content_2nic));
    if(ret0 != hipSuccess){
        printf("Error on allocation post content!\n");
        return -1;
    }
    ret0 = hipMemcpy(d_post2, post_cont2, sizeof(struct server_content_2nic), hipMemcpyHostToDevice);
    if(ret0 != hipSuccess){
        printf("Error on poll copy!\n");
        return -1;
    }

    // hipSetDevice(0);
    alloc_content<<<1,1>>>(d_post, d_poll);
    alloc_global_content<<<1,1>>>(d_post, d_poll, d_post2, gpu_mem);
    ret0 = hipDeviceSynchronize();
    if(ret0 != hipSuccess){
        printf("Error on alloc_content!\n");
        return -1;
    }
    return 0;
}

__device__ size_t sum_page_faults = 0;

__global__ void
print_retires(void){
    // size_t max = cq_wait[0];
    // for (size_t i = 0; i < 128; i++)
    // {
    //     if(max < cq_wait[i]) max = cq_wait[i];
    // }
    sum_page_faults += g_qp_index;
    printf("g_qp_index: %llu sum page fault: %llu\n", g_qp_index, sum_page_faults);
    g_qp_index = 0;
    // for (size_t i = 0; i < 128; i++)
    // {
    //     max = 0;
    // }
}

__global__ __launch_bounds__(1024,2) 
void
calculate_opt(size_t n, size_t size, rdma_buf<unsigned int> *rdma_array, unsigned int *array) {


    // Page size in elements (64KB / 4 bytes per unsigned int)
    const size_t pageSize = REQUEST_SIZE/4 / sizeof(unsigned int);
    // Elements per warp
    const size_t elementsPerWarp = pageSize / warpSize;

    // Global thread ID
    size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    // if(tid == 0) printf("warpSize: %d\n", warpSize);
    // Warp ID within the block
    size_t warpId = tid / warpSize;

    // Thread lane within the warp
    size_t lane = threadIdx.x % warpSize;

    // Determine which page this warp will process
    size_t pageStart = warpId * pageSize;

    // Ensure we don't process out-of-bounds pages
    if (pageStart < n * pageSize) {
        
        // Process elements within the page
        // for (size_t i = 0; i < elementsPerWarp; ++i) {
        //     size_t elementIdx = pageStart + lane + i * warpSize;
            uint end = (warpId + 1)*pageSize > size ? size : (warpId + 1)*pageSize;
            for(size_t j = warpId*pageSize + lane; j < end; j += warpSize) {
                uint end_edge = (*rdma_array)[j]; // shared_data[j - pageStart];
                array[j] = end_edge;
            }
        // }
    }
}


__global__ void transfer_opt(size_t n, rdma_buf<unsigned int> *rdma_array) {


    // Page size in elements (64KB / 4 bytes per unsigned int)
    const size_t pageSize = 4*1024 / sizeof(unsigned int);
    // Elements per warp
    const size_t elementsPerWarp = pageSize / warpSize;

    // Global thread ID
    size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    // if(tid == 0) printf("warpSize: %d\n", warpSize);
    // Warp ID within the block
    size_t warpId = tid / warpSize;

    // Thread lane within the warp
    size_t lane = threadIdx.x % warpSize;

    // Determine which page this warp will process
    size_t pageStart = warpId * pageSize;

    // Ensure we don't process out-of-bounds pages
    if (pageStart < n * pageSize) {
        
        // Process elements within the page
        // for (size_t i = 0; i < elementsPerWarp; ++i) {
        //     size_t elementIdx = pageStart + lane + i * warpSize;
                
            for(size_t j = warpId*pageSize + lane; j < (warpId + 1)*pageSize; j += warpSize) {
                uint end_edge = (*rdma_array)[j]; // shared_data[j - pageStart];
            }
        // }
    }
}

void compute_benchmark(){

    hipError_t ret;
    unsigned int *cuda_array, *h_cuda_array;
    uint64_t numblocks_update, numthreads, numblocks_kernel;
    double avg_milliseconds;
    float milliseconds;
    size_t num_elements, size = 12*1024*1024*1024llu; 
    num_elements = size/sizeof(uint);
    hipEvent_t start, end;

    rdma_buf<unsigned int> *rdma_array;
    check_cuda_error(hipMallocManaged((void **) &rdma_array, sizeof(rdma_buf<unsigned int>)));
    rdma_array->start(num_elements *sizeof(unsigned int), GPU, NULL);
    for(size_t i = 0; i < num_elements; i++){
        rdma_array->local_buffer[i] = 14;
    }

    check_cuda_error(hipEventCreate(&start));
    check_cuda_error(hipEventCreate(&end));

    // h_cuda_array = new uint[num_elements];
    auto start_chrono = std::chrono::steady_clock::now();
    check_cuda_error(hipHostMalloc((void **)&h_cuda_array, size));
    for(size_t i = 0; i < num_elements; i++){
        h_cuda_array[i] = 9;
    }
    check_cuda_error(hipMalloc((void **) &cuda_array, size));
    hipDeviceSynchronize();
    
    check_cuda_error(hipEventRecord(start, (hipStream_t) 1));

    check_cuda_error(hipMemcpy(cuda_array, h_cuda_array, size, hipMemcpyHostToDevice));
    
    check_cuda_error(hipEventRecord(end, (hipStream_t) 1));
    hipDeviceSynchronize();
    auto end_chrono = std::chrono::steady_clock::now();
    
    
    check_cuda_error(hipEventSynchronize(start));
    check_cuda_error(hipEventSynchronize(end));
    check_cuda_error(hipEventElapsedTime(&milliseconds, start, end));

    long duration = std::chrono::duration_cast<std::chrono::milliseconds>(end_chrono - start_chrono).count();
    printf("ret: %d hipGetLastError: %d\n", ret, hipGetLastError());
    printf("Elapsed time for hipMemcpy in milliseconds : %li ms bw: %.2f\n\n", duration, (float) size/(duration*0.001*1024*1024*1024) );
    printf("CUDA elapsed time for hipMemcpy in milliseconds : %.2f ms bw: %.2f\n\n", milliseconds, (float) size/(milliseconds*0.001*1024*1024*1024) );

    // numblocks_update = ((numVertex + numthreads) / numthreads);
    dim3 blockDim_kernel(numthreads, (numblocks_kernel+numthreads)/numthreads);
    // dim3 blockDim_kernel(BLOCK_SIZE, (numblocks_kernel+BLOCK_SIZE)/BLOCK_SIZE);
    numthreads = BLOCK_SIZE;
    // numblocks_update = ((numVertex + numthreads) / numthreads);
    dim3 blockDim_update(BLOCK_SIZE, (numblocks_update+BLOCK_SIZE)/BLOCK_SIZE);

    avg_milliseconds = 0.0f;

    ret = hipDeviceSynchronize();
    printf("ret: %d hipGetLastError: %d\n", ret, hipGetLastError());
    start_chrono = std::chrono::steady_clock::now();
    printf("starting kernel\n");
    size_t n_pages = size/(REQUEST_SIZE/4); // (REQUEST_SIZE);

    check_cuda_error(hipEventRecord(start, (hipStream_t) 1));
    numthreads = 1024;
<<<<<<< HEAD
    calculate_opt<<< (n_pages*32)/numthreads+1, numthreads >>>(num_elements, num_elements, rdma_array, cuda_array);
=======
    calculate_opt<<<(n_pages*32)/numthreads+1, numthreads>>>(num_elements, num_elements, rdma_array, cuda_array);
>>>>>>> origin/cloudlab
    check_cuda_error(hipEventRecord(end, (hipStream_t) 1));

    ret = hipDeviceSynchronize();               
    end_chrono = std::chrono::steady_clock::now();
    duration = std::chrono::duration_cast<std::chrono::milliseconds>(end_chrono - start_chrono).count();
    print_retires<<<1,1>>>();
    printf("ret: %d hipGetLastError: %d\n", ret, hipGetLastError());
    printf("Elapsed time in milliseconds : %li ms\n\n", duration);
    // printf("ret: %d hipGetLastError: %d\n", ret, hipGetLastError());
    
    check_cuda_error(hipMemcpy(h_cuda_array, cuda_array, size, hipMemcpyDeviceToHost));
    size_t num_errs = 0;
    for (size_t i = 0; i < num_elements; i++)
    {
        if(h_cuda_array[i] != 14){
            num_errs++;
            // printf("i: %d h_cuda_array[i]: %d\n", i, h_cuda_array[i]);
        }
    }
    

    
    check_cuda_error(hipEventSynchronize(start));
    check_cuda_error(hipEventSynchronize(end));
    check_cuda_error(hipEventElapsedTime(&milliseconds, start, end));
    printf("CUDA elapsed time in milliseconds : %0.3f ms num_errs: %llu bw: %.2f REQUEST_SIZE: %d\n\n", milliseconds, num_errs, (float) size/(milliseconds*0.001*1024*1024*1024), REQUEST_SIZE/1024);

}

void transfer_benchmark(){
    hipError_t ret;
    
    uint64_t numblocks_update, numthreads, numblocks_kernel;
    double avg_milliseconds;
    float milliseconds;
    size_t num_elements, size = 1*1024*1024*1024llu; 
    num_elements = size/sizeof(uint);
    
    hipEvent_t start, end;

    rdma_buf<unsigned int> *rdma_array;
    check_cuda_error(hipMallocManaged((void **) &rdma_array, sizeof(rdma_buf<unsigned int>)));

    rdma_array->start(num_elements *sizeof(unsigned int), GPU, NULL);

    for(size_t i = 0; i < num_elements; i++){
        rdma_array->local_buffer[i] = 14;
    }

   
    check_cuda_error(hipEventCreate(&start));
    check_cuda_error(hipEventCreate(&end));
   

    // numblocks_update = ((numVertex + numthreads) / numthreads);

    dim3 blockDim_kernel(numthreads, (numblocks_kernel+numthreads)/numthreads);
    // dim3 blockDim_kernel(BLOCK_SIZE, (numblocks_kernel+BLOCK_SIZE)/BLOCK_SIZE);
    numthreads = BLOCK_SIZE;
    // numblocks_update = ((numVertex + numthreads) / numthreads);
    dim3 blockDim_update(BLOCK_SIZE, (numblocks_update+BLOCK_SIZE)/BLOCK_SIZE);

    avg_milliseconds = 0.0f;

    printf("Initialization done\n");
    fflush(stdout);

    
    ret = hipDeviceSynchronize();
    printf("ret: %d hipGetLastError: %d\n", ret, hipGetLastError());
    

    auto start_chrono = std::chrono::steady_clock::now();
    
    printf("starting kernel\n");
    size_t n_pages = size/(4*1024);

    check_cuda_error(hipEventRecord(start, (hipStream_t) 1));
    transfer_opt<<<(n_pages*32)/512+1, 512>>>(num_elements, rdma_array);
    check_cuda_error(hipEventRecord(end, (hipStream_t) 1));

    ret = hipDeviceSynchronize();               
    auto end_chrono = std::chrono::steady_clock::now();
    long duration = std::chrono::duration_cast<std::chrono::milliseconds>(end_chrono - start_chrono).count();
    print_retires<<<1,1>>>();
    printf("ret: %d hipGetLastError: %d\n", ret, hipGetLastError());
    printf("Elapsed time in milliseconds : %li ms\n\n", duration);
    // printf("ret: %d hipGetLastError: %d\n", ret, hipGetLastError());
    

    
    check_cuda_error(hipEventSynchronize(start));
    check_cuda_error(hipEventSynchronize(end));
    check_cuda_error(hipEventElapsedTime(&milliseconds, start, end));
    printf("CUDA elapsed time in milliseconds : %li ms\n\n", milliseconds);

}

// Main program
int main(int argc, char **argv)
{   
    hipSetDevice(GPU);

    // init_gpu(0);
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    printf("deviceCount: %d\n", deviceCount);

    bool rdma_flag = true;
    hipError_t ret1;
    struct context_2nic *s_ctx = (struct context_2nic *)malloc(sizeof(struct context_2nic));
    if(rdma_flag){
        s_ctx->gpu_cq = NULL;
        s_ctx->wqbuf = NULL;
        s_ctx->cqbuf = NULL;
        s_ctx->gpu_qp = NULL;


        int num_msg = (unsigned long) atoi(argv[4]);
        int mesg_size = (unsigned long) atoi(argv[5]);
        int num_bufs = (unsigned long) atoi(argv[6]);

        
        struct post_content post_cont, *d_post, host_post;
        struct poll_content poll_cont, *d_poll, host_poll;
        // struct post_content2 /*post_cont2,*/ *d_post2;
        struct server_content_2nic post_cont2, *d_post2;
        struct host_keys keys;
        struct gpu_memory_info gpu_mem;

        int num_iteration = num_msg;
        s_ctx->n_bufs = num_bufs;

        s_ctx->gpu_buf_size = 12*1024*1024*1024llu; // N*sizeof(int)*3llu;
        s_ctx->gpu_buffer = NULL;

        // // remote connection:
        // int ret = connect(argv[2], s_ctx);

        // local connect
        char *mlx_name = "mlx5_0";
        // int ret = local_connect(mlx_name, s_ctx);
        int ret = local_connect_2nic(mlx_name, s_ctx, 0, GPU);

        mlx_name = "mlx5_3";
        // int ret = local_connect(mlx_name, s_ctx);
        ret = local_connect_2nic(mlx_name, s_ctx, 1, GPU);

        ret = prepare_post_poll_content_2nic(s_ctx, &post_cont, &poll_cont, &post_cont2, \
                                        &host_post, &host_poll, &keys, &gpu_mem);
        if(ret == -1) {
            printf("Post and poll contect creation failed\n");    
            exit(-1);
        }

        printf("alloc synDev ret: %d\n", hipDeviceSynchronize());
        hipSetDevice(GPU);
        alloc_global_cont(&post_cont, &poll_cont, &post_cont2, gpu_mem);

        // if(hipSuccess != ){    
        printf("alloc synDev ret1: %d\n", hipDeviceSynchronize());
            // return -1;
        // }

        ret1 = hipDeviceSynchronize();
        printf("ret: %d\n", ret1);
        if(hipSuccess != ret1){    
            return -1;
        }

        size_t restricted_gpu_mem = 16*1024*1024*1024llu;
        // restricted_gpu_mem = restricted_gpu_mem / 3;
        const size_t page_size = REQUEST_SIZE;
        // const size_t numPages = ceil((double)restricted_gpu_mem/page_size);

        printf("function: %s line: %d\n", __FILE__, __LINE__);
        alloc_global_host_content(host_post, host_poll, keys, gpu_mem);
        printf("function: %s line: %d\n", __FILE__, __LINE__);

        ret1 = hipDeviceSynchronize();
        printf("ret: %d\n", ret1);
        if(hipSuccess != ret1){    
            return -1;
        }

        ret1 = hipDeviceSynchronize();
        printf("ret: %d\n", ret1);
        if(hipSuccess != ret1){    
            return -1;
        }
        
        printf("restricted_gpu_mem: %zu\n", restricted_gpu_mem);
        hipSetDevice(GPU);
        start_page_queue<<<1, 1>>>(/*s_ctx->gpu_buf_size*/restricted_gpu_mem, page_size);
        ret1 = hipDeviceSynchronize();
        printf("ret: %d\n", ret1);
        if(hipSuccess != ret1){    
            return -1;
        }
    }

    
    

    if(rdma_flag){
        compute_benchmark();
        // transfer_benchmark();
        hipFree(s_ctx->gpu_buffer);
    }

    
    // printf("oversubs ratio: %d\n", oversubs_ratio_macro-1);
    
	return 0;
}